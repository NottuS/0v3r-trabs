#include "hip/hip_runtime.h"
#include "gpuIntegral.h"
#include "gpuUtils.h"
#include <stdio.h>
#include <stdlib.h>

__global__ void rowsSum(float *data, float *r_data2, int height, int width) {
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	if (row < height && col < width) {
		r_data2[row*width + col] = 0;
		for (int i = col - 1; i >= 0; i--)
			r_data2[row*width + col] += data[row * width + i];
	}
}

__global__ void colsSum(float *data, float *r_data1, int height, int width){
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	
	if (row < height && col < width) {
		r_data1[row*width + col] = 0;
		for (int i = row - 1; i >= 0; i--)
			r_data1[row*width + col] += data[i * width + col];
	}
}

__global__ void matrixSum(float *data, float *r_data1, float *r_data2, int height, int width){
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	if (row < height && col < width) {
		r_data1[row*width + col] = data[row*width + col] + r_data1[row*width + col] + r_data2[row*width + col];
	}
}

__global__ void integralImage(float *r_data1, float *r_data2, int height, int width){
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	if (row < height && col < width) {
		r_data2[row*width + col] = r_data1[row*width + col];
		for (int i = 1; i <= row && i <= col; i++)
			r_data2[row*width + col] +=  r_data1[(row - i)*width + col - i];
	}
}

void gpuIntegralImage(float *data, float *r_data, int height, int width){
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid(width / dimBlock.x, height / dimBlock.y);
	/*int dimBlock = BLOCK_START_SIZE;
	int dimGrid = ceil(height * width / (float) dimBlock);*/
	float *d_data = NULL; float *d_r_data1 = NULL; float *d_r_data2 = NULL;
	float *teste = (float *)calloc(height * width, sizeof(float));

	CUDA_CHECK_RETURN(hipSetDevice(0));
	CUDA_CHECK_RETURN(hipMalloc((void **)&d_data, height * width * sizeof(float))) ;
	CUDA_CHECK_RETURN(hipMalloc((void **)&d_r_data1, height * width * sizeof(float)));
	CUDA_CHECK_RETURN(hipMalloc((void **)&d_r_data2, height * width * sizeof(float)));
	CUDA_CHECK_RETURN(hipMemcpy(d_data, data, height * width * sizeof(float), hipMemcpyHostToDevice));
	
	rowsSum <<<dimGrid, dimBlock>>>(d_data, d_r_data1, height, width);
	for (int i = 0; i < height; i++) {
		//teste[i*width + j] = 0;
		for (int j = 1; j < width; j++)
		{
			teste[i*width + j] += data[i*width + j - 1] + teste[i*width + j - 1];
		}
	}

	colsSum <<<dimGrid, dimBlock>>>(d_data, d_r_data2, height, width);
	CUDA_CHECK_RETURN(hipDeviceSynchronize());	// Wait for the GPU launched work to complete
	matrixSum <<<dimGrid, dimBlock>>>(d_data, d_r_data1, d_r_data2, height, width);
	CUDA_CHECK_RETURN(hipDeviceSynchronize());	// Wait for the GPU launched work to complete
	integralImage <<<dimGrid, dimBlock>>>(d_r_data1, d_r_data2, height, width);
	CUDA_CHECK_RETURN(hipDeviceSynchronize());	// Wait for the GPU launched work to complete

	CUDA_CHECK_RETURN(hipMemcpy(r_data, d_r_data2, height * width * sizeof(float), hipMemcpyDeviceToHost)); //<--CORRETO
	//CUDA_CHECK_RETURN(hipMemcpy(r_data, d_r_data1, height * width * sizeof(float), hipMemcpyDeviceToHost));// <--- TESTE
	/*for (int i = 0; i < height * width; i++) {
		if (teste[i] != r_data[i])
			int y;//return FALSE;
	}*/
	hipFree(d_data);
	hipFree(d_r_data1);
	hipFree(d_r_data2);
}
