#include "hip/hip_runtime.h"

#include "matrix.h"
#include "EKF.h"

float *odometryError(const float *control, int dim){
	float *error = (float *) calloc(dim * dim, sizeof(float));

	error[0] = control[CONTROL_X] * 0.2;
	error[dim + 1] = control[CONTROL_Y] * 0.2;
	error[2*dim + 2] = control[CONTROL_TETA] * 0.2;

	if( error[2*dim + 2]> M_PI ){
		error[2*dim + 2] -= 2*M_PI;
	}

	if( error[2*dim + 2] < -M_PI ){
		error[2*dim + 2] += 2*M_PI;
	}

	return error;
}

float *moveUpadate(float *mean, const float *control) {
	mean[MEAN_X] = mean[MEAN_X] + control[CONTROL_X] * cos(mean[MEAN_TETA] + control[CONTROL_TETA]/2);
	mean[MEAN_Y] = mean[MEAN_Y] + control[CONTROL_Y] * sin(mean[MEAN_TETA] + control[CONTROL_TETA]/2);
	mean[MEAN_TETA] = mean[MEAN_TETA] + control[CONTROL_TETA];

	if( mean[MEAN_TETA] > M_PI ){
		mean[MEAN_TETA] -= 2*M_PI;
	}

	if( mean[MEAN_TETA] < -M_PI ){
		mean[MEAN_TETA] += 2*M_PI;
	}

	return mean;
}

// check if right
float *jacobianG(float teta, const float *control, int dim){
	float *G = (float *) calloc(dim * dim, sizeof(float));

	sCreateIdentity(G, dim);
	G[2] = -sin(teta + control[CONTROL_TETA]/2);
	G[dim + 2] = cos(teta + control[CONTROL_TETA]/2);

	return G;
}

//TODO
float *observationError(const float *observation, int dim) {
	float *error = (float *) calloc(dim * dim, sizeof(float));

	return error;
}

float *jacobianH(float *mean, const float *observation, int dim){
	float *H = (float *) calloc(dim * 2, sizeof(float));
	int landmark = observation[LM_IND] * 2 + 3;
	float x = mean[landmark] - mean[MEAN_X];
	float y = mean[landmark + 1] - mean[MEAN_X];
	float d2 = x*x - y*y;
	float d = sqrt(d2);

	H[0] = -x/d;
	H[1] = -y/d;
	H[2] = 0;
	H[dim] = y/d2;
	H[dim + 1] = -x/d2;
	H[dim + 2] = -1;
	H[landmark] = x/d;
	H[landmark + 1] = y/d;
	H[dim + landmark] = - y/d2;
	H[dim + landmark + 1] = x/d2;

	return H;
}

float *getExpected(float *mean, const float *observation, int dim){
	float *expected = (float *) calloc(dim, sizeof(float));
	int landmark = observation[LM_IND] * 2 + 3;
	float x = mean[landmark] - mean[MEAN_X];
	float y = mean[landmark + 1] - mean[MEAN_X];
	float d = sqrt( x*x - y*y);

	expected[landmark] = observation[FOCAL_LENGTH] * observation[K_X] * (observation[CAM_X]/observation[CAM_Z]);
	expected[landmark + 1] = observation[FOCAL_LENGTH] * observation[K_Y] * (observation[CAM_Y]/observation[CAM_Z]);
	expected[landmark] -= d;
	expected[landmark + 1] -= atan2(x,y) + mean[MEAN_TETA];

	if( expected[landmark + 1] > M_PI ){
		expected[landmark + 1] -= 2*M_PI;
	}

	if( expected[landmark + 1] < -M_PI ){
		expected[landmark + 1] += 2*M_PI;
	}

	return expected;
}

bool landmarkExist(const float *observation) {
	if(observation[L_EXISTS] < 0)
		return false;
	return true;
}

void addLandmark(float *mean, float *covariance, const float *observation, int *dim){
	*dim = *dim + 2;
	mean = (float *) realloc(mean, (*dim) * sizeof(float));
	covariance = (float *) realloc(covariance, sizeof(float) * (*dim) * (*dim));

	//Set the position(x,y) of the landmark in relation to the robot
	mean[*dim - 2] = mean[MEAN_X] + observation[FOCAL_LENGTH] *
			observation[K_X] * (observation[CAM_X]/observation[CAM_Z]);
	mean[*dim - 1 ] = mean[MEAN_Y] + observation[FOCAL_LENGTH] *
			observation[K_Y] * (observation[CAM_Y]/observation[CAM_Z]);
}


//TODO take care on mul transpose, the lenghs of the matrix must be swaped
void EKF(int dim, float *mean, float *covariance, const float *control, const float *observation){
	//******Update step******
	//u_t = g(control_t, mean_t-1)
	float teta = mean[MEAN_TETA];
	moveUpadate(mean, control);

	//TODO this block can be improved look the slides of stachness, and G is sparse
	float *partial = (float *)malloc(sizeof(float) * dim * dim);
	float *G = jacobianG(teta, control, dim);
	float *temp;
	//G * E_t-1
	sMatMul(NOT_TRANSP, NOT_TRANSP, partial, G, covariance, dim, dim, dim);
	//TODO take care on mul transpose, the lenghs of the matrix must be swaped
	//G * E_t-1 * G^T
	sMatMul(NOT_TRANSP, TRANSP, covariance, partial, G, dim, dim, dim);
	temp = odometryError(control, dim);
	//E = G * E_t-1 * G^T + OdometryError_t
	sMatSum(covariance, covariance, temp, dim, dim);
	free(G);
	free(temp);


	////******Matching and compute h(observation_t)******
	if(!landmarkExist(observation)){
		addLandmark(mean, covariance, observation, &dim);
		partial = (float *) realloc(partial, sizeof(float) * dim * dim);
	}

	//******Correction/Update step******
	//H is sparse, so this block can be improved...
	float *kalmanGain = (float *) malloc(sizeof(float) * dim * dim);
	float *H = jacobianH(mean, observation, dim);
	//E * H^T
	sMatMul(NOT_TRANSP, TRANSP, partial, H, covariance, dim, dim, dim);
	//H * E * H^T
	sMatMul(NOT_TRANSP, NOT_TRANSP, kalmanGain, H, partial, dim, dim, dim);
	//(H * E * H^T + observationError)
	temp = observationError(observation, dim);
	sMatSum(temp, kalmanGain, temp, dim, dim);
	//(H * E * H^T + observationError)^-1
	//check this, probabily wrong
	sMatInverse(temp, dim, dim);
	//K = E * H^T * (H * E * H^T + observationError)^-1
	sMatMul(NOT_TRANSP, NOT_TRANSP, kalmanGain, partial, temp, dim, dim, dim);

	//z - h(u)
	float *expected = (float *) getExpected(mean, observation, dim);
	float *temp2 =  (float *) malloc(sizeof(float) * dim);
	//K(z - h(u))
	sMatMul(NOT_TRANSP, NOT_TRANSP, temp2, kalmanGain, expected, dim, dim, 1);
	//u = u + K(z - h(u))
	sMatSum(mean, mean, temp2, dim, 1);
	free(expected);
	free(temp2);
	//K * H
	sMatMul(NOT_TRANSP, NOT_TRANSP, partial, kalmanGain, H, dim, dim, dim);
	//(I - K * H)
	//temp = Identity matrix
	memset(temp, 0, sizeof(float));
	sCreateIdentity(temp, dim);
	//THIS can be improved...
	sMatSub(partial, temp, partial, dim, dim);
	free(kalmanGain);
	free(H);

	//E = (I - K * H) * E
	sMatMul(NOT_TRANSP, NOT_TRANSP, temp, partial, covariance, dim, dim, dim);
	free(covariance);
	covariance = temp;
	free(temp);
	free(partial);
}
