#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include "matrix.h"
#include <math.h>

#define BLOCk 32

void choleskyDecomp(const float *A, float *L, int n, int nr_cols_A){
	int i,j,k;
	float sum;
	for(i = 0; i < n; i++) {
		for (j = 0; j <= i; ++j) {
			sum = A[i*n + j];
			for(k = 0; k < j; k++) {
				sum -= L[i*n +k] * L[j*n + k];
			}
			if(i == j) {
				L[i*n + j] = sqrt(sum);
			} else {
				L[i*n + j] = sum / L[j*n + j];
			}
		}
	}
}

void choleskyDecompositionGPU( int n, float *cpu_A, float * d_A, float *d_L){
	for (int i = 0; i < n; i += BLOCk) {
		int h = n - i;
		int w = h < BLOCk ? h : BLOCk
		if(i > 0){
			srku();
			mul();
			if(h > BLOCk){
				srku();
			}
		}

		choleskyDecomp();
		if(h > BLOCk){
			solve();
		}
	}
}

void invert( int n) {
	thrust::device_vector<float> d_A(n * n);
	thrust::device_vector<float> d_L(n * n);
	thrust::host_vector<float>h_A(n * n);
	thrust::host_vector<float>h_L(n * n);
	float A = thrust::raw_pointer_cast(&h_A[0]);

	srand(n);
	for( int i = 0; i < n; i++ ) {
		for (int j = i; j < n; j++) {
			A[i*n+j] = 2.0*(rand()%32768)/32768.0;
			A[j*n+i] = A[i*n+j];
		}
		A[i*n+i] += 2*sqrt((float)n);
	}


}
