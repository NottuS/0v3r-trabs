#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include "matrix.h"
#include <math.h>
#include "Inverse.h"

void fowardSubst(float *res, float *L, float *I, int n){

	for(int i = 0; i < n; i++){
		for(int j = 0; j < n; j++){
			float sum = I[j*n+i];
			for(int k = 0; k < j; k++){
				sum -= L[j*n + k]*res[k*n+i];
			}
			res[j*n + i] = sum/L[j*n+j];
		}
	}
}

__global__ void fowardSubstKernel(float *res, float *L, float *I, int n){
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	if (col < n) {
		for(int j = 0; j < n; j++){
			float sum = I[j*n+col];
			for(int k = 0; k < j; k++){
				sum -= L[j*n + k]*res[k*n+col];
			}
			res[j*n + col] = sum/L[j*n+j];
		}
	}
}

void backSubst(float *res, float *L, float *I, int n){
	for(int i = 0; i < n; i++){
		for(int j = n-1; j >= 0; j--){
			float sum = I[j*n+i];
			for(int k = n - 1; k > j; k--){
				sum -= L[j*n + k]*res[k*n+i];
			}
			res[j*n + i] = sum/L[j*n+j];
		}
	}
}

__global__ void  backSubstkernel(float *res, float *L, float *I, int n){
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	if (col < n) {
		for(int j = n-1; j >= 0; j--){
			float sum = I[j*n+col];
			for(int k = n - 1; k > j; k--){
				sum -= L[j*n + k]*res[k*n+col];
			}
			res[j*n + col] = sum/L[j*n+j];
		}
	}
}

void choleskyDecomp(float *L, const float *A, int n){
	int i,j,k;
	float sum;
	for(i = 0; i < n; i++) {
		for (j = 0; j <= i; ++j) {
			sum = A[i*n + j];
			for(k = 0; k < j; k++) {
				sum -= L[i*n +k] * L[j*n + k];
			}
			if(i == j) {
				L[i*n + j] = sqrt(sum);
			} else {
				L[i*n + j] = sum / L[j*n + j];
			}
		}
	}
}

__global__ void choleskyDecompKernel2(int ind, const float *A, float *L, int n){
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	__shared__ float tempA[BLOCK_SIZE][BLOCK_SIZE];
	__shared__ float tempB[BLOCK_SIZE][BLOCK_SIZE];

	if (row < n && col < BLOCK_SIZE) {
		//TODO Check if all their work is done, and the indexs
		float sum = A[row * n + ind];
			for(int i = col; i < ind; i += BLOCK_SIZE){
				sum -= L[ind * n + i] * L[row * n + i];
			}



		/*for (int i = 0; i < ceil(float(ind)/(BLOCK_SIZE)); ++i) {

			/*tempA[threadIdx.y*BLOCK_SIZE + threadIdx.x] = A[row * nr_col_A + i * BLOCK_START_SIZE + threadIdx.x];
			tempB[threadIdx.y*BLOCK_SIZE + threadIdx.x] = B[(i * BLOCK_START_SIZE + threadIdx.x)* nr_cols_B *  + row];
			tempA[threadIdx.y][threadIdx.x] = A[ind * nr_cols_A + BLOCK_SIZE * i + threadIdx.x];
			tempB[threadIdx.y][threadIdx.x] = A[row * nr_cols_A + BLOCK_SIZE * i + threadIdx.x];
			__syncthreads();
			if(ind > (i + 1) * BLOCK_SIZE){
				for (int j = 0; j < BLOCK_SIZE ; ++j)
					sum -= tempA[threadIdx.y][j] * tempB[threadIdx.y][j];
			} else {
				for (int j = 0; j < ind % BLOCK_SIZE ; ++j)
					sum -= tempA[threadIdx.y][j] * tempB[threadIdx.y][j];
			}
			__syncthreads();
		}*/
		L[row * n + col] = sum;
		__syncthreads();
		if( row == ind ){
			for(int i = 0; i < BLOCK_SIZE; i++){
				sum -= L[row * n + i];
			}
			sum = sqrtf(sum);
		}
	}
}

__global__ void choleskyDecompKernel(int ind, const float *A, float *L, int n){
	/*int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;*/
	int row = blockIdx.x * blockDim.x + threadIdx.x;

	__shared__ float temp1[BLOCK_START_SIZE];
	//__shared__ float temp2[BLOCK_START_SIZE];

	if (row < n) {
		int x = threadIdx.x;
		float sum = A[row * n + ind];
		for (int k = 0; k < ceilf((ind + 0.0)/BLOCK_START_SIZE); ++k) {
			temp1[x] = L[ind * n+ BLOCK_START_SIZE * k + x];
			//temp2[x] = L[row * nr_cols_A + BLOCK_START_SIZE * k + 0];

			__syncthreads( );
			if((k+1)*BLOCK_START_SIZE <= ind) {
				#pragma unroll
				for(int i = 0; i < BLOCK_START_SIZE; i += 8){
					float a,b,c,d, a1, b1, c1, d1;
					a = L[row * n + k * BLOCK_START_SIZE + i];
					b = L[row * n + k * BLOCK_START_SIZE + i + 1];
					c = L[row * n + k * BLOCK_START_SIZE + i + 2];
					d = L[row * n + k * BLOCK_START_SIZE + i + 3];
					a1 = L[row * n + k * BLOCK_START_SIZE + i + 4];
					b1 = L[row * n + k * BLOCK_START_SIZE + i + 5];
					c1 = L[row * n + k * BLOCK_START_SIZE + i + 6];
					d1 = L[row * n + k * BLOCK_START_SIZE + i + 7];
					//sum -= L[row * nr_cols_A + k * BLOCK_START_SIZE + i] * temp1[i];
					//sum -= temp2[i] * temp1[i];
					sum -= a * temp1[i] + b * temp1[i+1] + c * temp1[i+2] + d * temp1[i+3] +
							a1 * temp1[i+4] + b1 * temp1[i+5] + c1 * temp1[i+6] + d1 * temp1[i+7];
				}
			} else {
				for(int i = 0; i < ind % BLOCK_START_SIZE; i += 8){
					float a,b,c,d, a1, b1, c1, d1;
					a = L[row * n + k * BLOCK_START_SIZE + i];
					b = L[row * n + k * BLOCK_START_SIZE + i + 1];
					c = L[row * n + k * BLOCK_START_SIZE + i + 2];
					d = L[row * n + k * BLOCK_START_SIZE + i + 3];
					a1 = L[row * n + k * BLOCK_START_SIZE + i + 4];
					b1 = L[row * n + k * BLOCK_START_SIZE + i + 5];
					c1 = L[row * n + k * BLOCK_START_SIZE + i + 6];
					d1 = L[row * n + k * BLOCK_START_SIZE + i + 7];
					//sum -= L[row * nr_cols_A + k * BLOCK_START_SIZE + i] * temp1[i];
					//sum -= temp2[i] * temp1[i];
					sum -= a * temp1[i] + b * temp1[i+1] + c * temp1[i+2] + d * temp1[i+3] +
							a1 * temp1[i+4] + b1 * temp1[i+5] + c1 * temp1[i+6] + d1 * temp1[i+7];
				}
			}
			__syncthreads();
		}

		if( row == ind )
			sum = sqrtf(sum);
		L[row * n + ind] = sum;
	}
}

__global__ void oldCD(int ind, const float *A, float *L, int n){
	int row = blockIdx.x * blockDim.x + threadIdx.x;

	__shared__ float temp1[BLOCK_START_SIZE];
	//__shared__ float temp2[BLOCK_START_SIZE];

	if (row < n) {
		int x = threadIdx.x;
		float sum = A[row * n + ind];
		for (int k = 0; k < ceilf((ind + 0.0)/BLOCK_START_SIZE); ++k) {
			temp1[x] = L[ind * n + BLOCK_START_SIZE * k + x];
						//temp2[x] = L[row * nr_cols_A + BLOCK_START_SIZE * k + 0];

			__syncthreads( );
			if((k+1)*BLOCK_START_SIZE <= ind) {
				#pragma unroll
				for(int i = 0; i < BLOCK_START_SIZE; i++){
					sum -= L[row * n+ k * BLOCK_START_SIZE + i] * temp1[i];
				}
			} else {
				for(int i = 0; i < ind % BLOCK_START_SIZE; i++){
					sum -= L[row * n+ k * BLOCK_START_SIZE + i] * temp1[i];
				}
			}
			__syncthreads();
		}
		if( row == ind )
			sum = sqrtf(sum);
		L[row * n + ind] = sum;
	}
}

__global__ void updateCholesky(int ind, float *L, int n){
	int row = blockIdx.x * blockDim.x + threadIdx.x;

	if (row < n && row != ind ) {
		if( row < ind )
			L[row * n + ind] = 0;
		else
			L[row * n+ ind] /=  L[ind * n + ind];
	}
}

//TODO
void pMatInverse(float *A, float *L, float *I, int n){
	dim3 dimBlock(BLOCK_START_SIZE);
//	dim3 dimBlock2(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid(ceil(float(n) / dimBlock.x));
	//dim3 dimGrid2(ceil(float(n) / dimBlock.y), BLOCK_SIZE);

	for(int i = 0; i < n; i++){
		choleskyDecompKernel<<<dimGrid, dimBlock>>>(i, A, L, n);
		CUDA_CHECK_RETURN(hipDeviceSynchronize());
		updateCholesky<<<dimGrid, dimBlock>>>(i, L, n);
		CUDA_CHECK_RETURN(hipDeviceSynchronize());
	}

	fowardSubstKernel<<<dimGrid, dimBlock>>>(A, L, I, n);
	CUDA_CHECK_RETURN(hipDeviceSynchronize());
	pMatTranspose(I, L, n, n);
	CUDA_CHECK_RETURN(hipDeviceSynchronize());
	backSubstkernel<<<dimGrid, dimBlock>>>(L, I, A, n);
	CUDA_CHECK_RETURN(hipDeviceSynchronize());

}

/*void choleskyDecompositionGPU( int n, float *cpu_A, float * d_A, float *d_L){
	for (int i = 0; i < n; i += BLOCk) {
		int h = n - i;
		int w = h < BLOCk ? h : BLOCk
		if(i > 0){
			srku();
			mul();
			if(h > BLOCk){
				srku();
			}
		}

		choleskyDecomp();
		if(h > BLOCk){
			solve();
		}
	}
}*/



void testInvert( int n) {
	thrust::device_vector<float> d_A(n * n);
	thrust::device_vector<float> d_L(n * n);
	thrust::device_vector<float> d_I(n * n);
	thrust::host_vector<float>h_A(n * n);
	thrust::host_vector<float>h_L(n * n);
	thrust::host_vector<float>h_I(n*n);
	thrust::host_vector<float>b_A(n*n);

	float *A = thrust::raw_pointer_cast(&h_A[0]);
	float *I = thrust::raw_pointer_cast(&h_I[0]);
	float *L = thrust::raw_pointer_cast(&h_L[0]);

	float *dA = thrust::raw_pointer_cast(&d_A[0]);
	float *dI = thrust::raw_pointer_cast(&d_I[0]);
	float *dL = thrust::raw_pointer_cast(&d_L[0]);

	srand(n);
	for( int i = 0; i < n; i++ ) {
		for (int j = i; j < n; j++) {
			A[i*n+j] = 2.0*(rand()%32768)/32768.0;
			A[j*n+i] = A[i*n+j];
		}
		A[i*n+i] += 2*sqrt((float)n);
	}

	thrust::fill(h_I.begin(), h_I.end(), 0);
	for(int i = 0; i < n; i++)
		I[i *n +i] = 1;
	thrust::copy(h_A.begin(), h_A.end(), b_A.begin());
	thrust::copy(h_A.begin(), h_A.end(), d_A.begin());
	thrust::copy(h_I.begin(), h_I.end(), d_I.begin());

	/*clock_t start = clock();
	choleskyDecomp(L, A, n);
	//print_matrix(L, n, n);
	fowardSubst(A, L, I, n);
	sMatTranspose(I, L, n, n);
	backSubst(L, I, A,  n);
	clock_t end = clock();
	printf("seq took: %f seconds \n", float(end - start) / CLOCKS_PER_SEC);*/
	//print_matrix(L, n, n);

	//print_matrix(L, n, n);


	thrust::copy(b_A.begin(), b_A.end(), h_A.begin());
	clock_t start = clock();
	sMatInverse(A, n,n , L);
	clock_t end = clock();
	//print_matrix(A, n, n);
	printf("seq2 took: %f seconds \n", float(end - start) / CLOCKS_PER_SEC);

	start = clock();
	pMatInverse(dA, dL, dI, n);
	end = clock();
	printf("par took: %f seconds \n", float(end - start) / CLOCKS_PER_SEC);
	thrust::copy(d_L.begin(), d_L.end(), h_A.begin());

}
