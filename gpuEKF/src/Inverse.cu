#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include "matrix.h"
#include <math.h>

#define BLOCk 32

void fowardSubst(float *L, float *I, float *res,float n){

	for(int i = 0; i < n; i++){
		for(int j = 0; j < n; j++){
			float sum = L[j*n+i];
			for(int k = 0; k < j; k++){
				sum -= L[j*n + k]*res[k*n+i];
			}
			res[j*n + i] = sum/L[j*n+j];
		}
	}
}

void backSubst(float *L, float *I, float *res,float n){
	for(int i = 0; i < n; i++){
		for(int j = n-1; j >= 0; j--){
			float sum = L[j*n+i];
			for(int k = n - 1; k > j; k--){
				sum -= L[j*n + k]*res[k*n+i];
			}
			res[j*n + i] = sum/L[j*n+j];
		}
	}
}

void choleskyDecomp(const float *A, float *L, int n){
	int i,j,k;
	float sum;
	for(i = 0; i < n; i++) {
		for (j = 0; j <= i; ++j) {
			sum = A[i*n + j];
			for(k = 0; k < j; k++) {
				sum -= L[i*n +k] * L[j*n + k];
			}
			if(i == j) {
				L[i*n + j] = sqrt(sum);
			} else {
				L[i*n + j] = sum / L[j*n + j];
			}
		}
	}
}

void choleskyDecompositionGPU( int n, float *cpu_A, float * d_A, float *d_L){
	for (int i = 0; i < n; i += BLOCk) {
		int h = n - i;
		int w = h < BLOCk ? h : BLOCk
		if(i > 0){
			srku();
			mul();
			if(h > BLOCk){
				srku();
			}
		}

		choleskyDecomp();
		if(h > BLOCk){
			solve();
		}
	}
}

void invert( int n) {
	thrust::device_vector<float> d_A(n * n);
	thrust::device_vector<float> d_L(n * n);
	thrust::host_vector<float>h_A(n * n);
	thrust::host_vector<float>h_L(n * n);
	float A = thrust::raw_pointer_cast(&h_A[0]);

	srand(n);
	for( int i = 0; i < n; i++ ) {
		for (int j = i; j < n; j++) {
			A[i*n+j] = 2.0*(rand()%32768)/32768.0;
			A[j*n+i] = A[i*n+j];
		}
		A[i*n+i] += 2*sqrt((float)n);
	}

	choleskyDecomp(h_A, h_L, n);
	fowardSubst(h_L, I, res,n);
	trasnpose(h_L,n);
	backSubst(h_L, res, I, n);

}
