/**
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 */
#include <stdio.h>
#include <stdlib.h>
#include<hipblas.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include"matrix.h"
#define IDX2C (i ,j , ld ) ((( j )*( ld ))+( i ))
#define m 6 // a - mxk matrix
#define n 4 // b - kxn matrix
#define k 5 // c - mxn matrix


/**
 * Host function that prepares data array and passes it to the CUDA kernel.
 */
int main(void) {
	hipblasHandle_t handle;
	hipblasCreate(&handle);

	// Allocate 3 arrays on CPU
	int nr_rows_A, nr_cols_A, nr_rows_B, nr_cols_B, nr_rows_C, nr_cols_C;

	// for simplicity we are going to use square arrays
	nr_rows_A = nr_cols_A = nr_rows_B = nr_cols_B = nr_rows_C = nr_cols_C = 3;

	thrust::device_vector<float> d_A(nr_rows_A * nr_cols_A),
			d_B(nr_rows_B * nr_cols_B), d_C(nr_rows_C * nr_cols_C);

	// Fill the arrays A and B on GPU with random numbers
	GPU_fill_rand(thrust::raw_pointer_cast(&d_A[0]), nr_rows_A, nr_cols_A);
	GPU_fill_rand(thrust::raw_pointer_cast(&d_B[0]), nr_rows_B, nr_cols_B);

	cublasMatMul(handle, thrust::raw_pointer_cast(&d_A[0]),
			thrust::raw_pointer_cast(&d_B[0]), thrust::raw_pointer_cast(&d_C[0]),
			nr_rows_A, nr_cols_A, nr_cols_B);
}
