#include "hip/hip_runtime.h"
/**
* Implement a parallel and a sequencial version of the EKF for SLAM.
* EKF(mean_t-1, covariance_t-1, control_t, observation_t)
* 
* 	mean'_t = g(control_t, mean_t-1)
* 	covariance'_t = Jacobian(g()) * covariance_t-1 * Jacobian(g())^T + OdometryError_t
* 
* 	KalmanGain = covariance'_t * Jacobian(h(observation_t)) * 
* 		(Jacobian(h(observation_t)) * covariance'_t * Jacobian(h(observation_t)) + ObservatioError_t)^-1
* 	mean_t = mean'_t + KalmanGain * (observation_t - h(observation_t))
* 	covariance_t = (I - KalmanGain* Jacobian(h(observation_t))) * covariance'_t
* 	return mean_t, covariance_t
 */
#include <stdio.h>
#include <stdlib.h>
#include <ctime>
#include <iostream>

#include<hipblas.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include"matrix.h"
#define m 6 // a - mxk matrix
#define n 4 // b - kxn matrix
#define k 5 // c - mxn matrix
#define TRANS 0
#define MUL 1
#define SUM 2
#define INV 3

void sumComp(hipblasHandle_t handle, thrust::host_vector<float>A, thrust::host_vector<float> B, thrust::host_vector<float> C,
		thrust::device_vector<float> d_A, thrust::device_vector<float>d_B,
		thrust::device_vector<float>d_C, int nr_rows_A, int nr_cols_A){

}

float *odometryError(float *control, int dim){
		
}

float *observationError(float *observation, int dim) {
	
}

float *jacobianG(int dim){
		
}

float *jacobianH(int dim){
		
}

float *moveUpadate(float *mean, float *control){
	int x = 0*3 + 0;
	int y = 1*3 + 1;
	int teta = 2*3 + 2;
	
	mean[x] = mean[x] + control[0] * cos(mean[teta] + control[2]/2);
	mean[y] = mean[y] + control[1] * sin(mean[teta] + control[2]/2);
	mean[teta] = media[teta] + control[2];
	
	return mean;
}

bool lanmarkExist(float *observation) {
	return true;
}

void addLandmark(float *mean, float *covariance, int *dim){
	
}

void EKF(int dim, float *mean, float *covariance, const float *control, const float *observation){
	//mean'_t = g(control_t, mean_t-1)
	float *mean = moveUpadate(mean, control);
	float *G = jacobianG(dim);
	
	//Update step
	float *partial = malloc(sizeof(float) * dim * dim);
	sMatMul(partial, G, covariance, dim, dim, dim);
	
	//vai dar bosta esse transpose tem q trocar o g
	sMatTranspose(G, G, dim, dim);
	sMatMul(covariance, partial, G, dim, dim, dim);
	sMatSum(covariance, covariance, odometryError(control, dim));

	//Matching and compute h(observation_t)
	if(!lanmarkExist(observation)){
		addLandmark(mean, covariance, &dim);
		partial = (float *) realloc(partial, sizeof(float) * dim * dim);
	}
	//compute h(observation_t)
	
	//Correction/update step
	float *kalmanGain = malloc(sizeof(float) * dim * dim);
	float H = jacobianH(dim);
	float H_t = malloc(sizeof(float) * dim * dim);
	
	sMatTranspose(H_t, H, dim, dim);
	sMatMul(partial, H_t, covariance);
	sMatMul(kalmanGain, H, partial);
	sMatSum(kalmanGain, kalmanGain, observationError(observation));
	//check this probabily wrong
	sMatMul(kalmanGain, Partial, kalmanGain);
	
	//missing compute mean
	
	sMatMul(partial, kalmanGain, H);
	//criar um special sub
	//sMatSub(partial, identity, partial);
	//check this probabily wrong
	sMatMul(covariance, partial, covariance);
}

void parallelEKF(float *mean, float *covariance){
	
}



int main(int argc, char** argv) {
	hipblasHandle_t handle;
	CUBLAS_CHECK_RETURN(hipblasCreate(&handle));
	int operation;
	// Allocate 3 arrays on CPU
	int nr_rows_A, nr_cols_A, nr_rows_B, nr_cols_B, nr_rows_C, nr_cols_C;

	if(argc < 3){
		nr_rows_A = nr_cols_A = nr_rows_B = nr_cols_B = nr_rows_C = nr_cols_C = 3;
		operation = 1;
	} else {
		nr_rows_A = nr_cols_A = nr_rows_B = nr_cols_B = nr_rows_C = nr_cols_C = atoi(argv[1]);
		operation = atoi(argv[2]);
	}

	thrust::device_vector<float> d_A(nr_rows_A * nr_cols_A),
			d_B(nr_rows_B * nr_cols_B), d_C(nr_rows_C * nr_cols_C);

	// Fill the arrays A and B on GPU with random numbers
	GPU_fill_rand(thrust::raw_pointer_cast(&d_A[0]), nr_rows_A, nr_cols_A);
	GPU_fill_rand(thrust::raw_pointer_cast(&d_B[0]), nr_rows_B, nr_cols_B);
	//thrust::device_vector<int> vec(d_A, d_A + n);

	thrust::host_vector<float>A(d_A.begin(), d_A.begin() + d_A.size());
	thrust::host_vector<float>B(d_B.begin(), d_B.begin() + d_B.size());
	thrust::host_vector<float>C(nr_rows_C * nr_cols_C);


	clock_t start = clock();
	switch (operation) {
		case TRANS:
			sMatTranspose(thrust::raw_pointer_cast(&C[0]),
					thrust::raw_pointer_cast(&A[0]), nr_rows_A, nr_cols_A);
			break;
		case MUL:
			sMatMul(thrust::raw_pointer_cast(&C[0]), thrust::raw_pointer_cast(&A[0]),
					thrust::raw_pointer_cast(&B[0]), nr_rows_A, nr_cols_A, nr_cols_A);
			break;
		case SUM:
			sMatSum(thrust::raw_pointer_cast(&C[0]), thrust::raw_pointer_cast(&A[0]),
					thrust::raw_pointer_cast(&B[0]), nr_rows_A, nr_cols_A);
			break;
		case INV:
			sMatInverse(thrust::raw_pointer_cast(&A[0]),
					nr_rows_A, nr_cols_A, thrust::raw_pointer_cast(&C[0]));
			break;
		default:
			break;
	}
	clock_t end = clock();
	printf("seq Mul took: %f seconds \n", float(end - start) / CLOCKS_PER_SEC);
	if(argc < 4 || atoi(argv[3]) != 0){
		print_matrix(thrust::raw_pointer_cast(&C[0]), nr_rows_A, nr_cols_A);
	}
	start = clock();
	switch (operation) {
		case TRANS:
			cublasMatTranspose(handle, thrust::raw_pointer_cast(&d_C[0]), thrust::raw_pointer_cast(&d_A[0]),
					thrust::raw_pointer_cast(&d_B[0]), nr_rows_A, nr_cols_A);
			break;
		case MUL:
			cublasMatMul(handle, thrust::raw_pointer_cast(&d_C[0]), thrust::raw_pointer_cast(&d_A[0]),
					thrust::raw_pointer_cast(&d_B[0]),	nr_rows_A, nr_cols_A, nr_cols_A);
			break;
		case SUM:
			cublasMatSum(handle, thrust::raw_pointer_cast(&d_C[0]), thrust::raw_pointer_cast(&d_A[0]),
					thrust::raw_pointer_cast(&d_B[0]), nr_rows_A, nr_cols_A);
			break;
		case INV:

			break;
		default:
			break;
	}

	end = clock();
	printf("par Mul took: %f seconds \n", float(end - start) / CLOCKS_PER_SEC);
	thrust::copy(d_C.begin(), d_C.end(), C.begin());
	// deallocate any capacity which may currently be associated with vec
	//vec.shrink_to_fit();

	if(argc < 4 || atoi(argv[3]) != 0){
		print_matrix(thrust::raw_pointer_cast(&C[0]), nr_rows_A, nr_cols_A);
	}

	start = clock();
	switch (operation) {
		case TRANS:
			pMatTranspose(thrust::raw_pointer_cast(&d_C[0]),
					thrust::raw_pointer_cast(&d_A[0]), nr_rows_A, nr_cols_A);
			break;
		case MUL:
			pMatMul(thrust::raw_pointer_cast(&d_C[0]), thrust::raw_pointer_cast(&d_A[0]),
					thrust::raw_pointer_cast(&d_B[0]),	nr_rows_A, nr_cols_A, nr_cols_A);
			break;
		case SUM:
			pMatSum(thrust::raw_pointer_cast(&d_C[0]), thrust::raw_pointer_cast(&d_A[0]),
					thrust::raw_pointer_cast(&d_B[0]), nr_rows_A, nr_cols_A);
			break;
		case INV:

			break;
		default:
			break;
	}

	end = clock();
	printf("My par Mul took: %f seconds \n", float(end - start) / CLOCKS_PER_SEC);
	thrust::copy(d_C.begin(), d_C.end(), C.begin());
	// deallocate any capacity which may currently be associated with vec
	//vec.shrink_to_fit();
	if(argc < 4 || atoi(argv[3]) != 0){
		print_matrix(thrust::raw_pointer_cast(&C[0]), nr_rows_A, nr_cols_A);
	}
	/*multComp(handle, A, B, C, d_A, d_B, d_C, nr_rows_A, nr_cols_A);
	sumComp(handle, A, B, C, d_A, d_B, d_C, nr_rows_A, nr_cols_A);*/
	hipblasDestroy(handle);
}
