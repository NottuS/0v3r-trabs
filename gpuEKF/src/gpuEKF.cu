#include "hip/hip_runtime.h"
/**
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 */
#include <stdio.h>
#include <stdlib.h>
#include <ctime>
#include <iostream>

#include<hipblas.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include"matrix.h"
#define m 6 // a - mxk matrix
#define n 4 // b - kxn matrix
#define k 5 // c - mxn matrix
#define TRANS 0
#define MUL 1
#define SUM 2
#define INV 3

void sumComp(hipblasHandle_t handle, thrust::host_vector<float>A, thrust::host_vector<float> B, thrust::host_vector<float> C,
		thrust::device_vector<float> d_A, thrust::device_vector<float>d_B,
		thrust::device_vector<float>d_C, int nr_rows_A, int nr_cols_A){

}

int main(int argc, char** argv) {
	hipblasHandle_t handle;
	CUBLAS_CHECK_RETURN(hipblasCreate(&handle));
	int operation;
	// Allocate 3 arrays on CPU
	int nr_rows_A, nr_cols_A, nr_rows_B, nr_cols_B, nr_rows_C, nr_cols_C;

	if(argc < 3){
		nr_rows_A = nr_cols_A = nr_rows_B = nr_cols_B = nr_rows_C = nr_cols_C = 3;
		operation = 1;
	} else {
		nr_rows_A = nr_cols_A = nr_rows_B = nr_cols_B = nr_rows_C = nr_cols_C = atoi(argv[1]);
		operation = atoi(argv[2]);
	}

	thrust::device_vector<float> d_A(nr_rows_A * nr_cols_A),
			d_B(nr_rows_B * nr_cols_B), d_C(nr_rows_C * nr_cols_C);

	// Fill the arrays A and B on GPU with random numbers
	GPU_fill_rand(thrust::raw_pointer_cast(&d_A[0]), nr_rows_A, nr_cols_A);
	GPU_fill_rand(thrust::raw_pointer_cast(&d_B[0]), nr_rows_B, nr_cols_B);
	//thrust::device_vector<int> vec(d_A, d_A + n);

	thrust::host_vector<float>A(d_A.begin(), d_A.begin() + d_A.size());
	thrust::host_vector<float>B(d_B.begin(), d_B.begin() + d_B.size());
	thrust::host_vector<float>C(nr_rows_C * nr_cols_C);


	clock_t start = clock();
	switch (operation) {
		case TRANS:
			sMatTranspose(thrust::raw_pointer_cast(&C[0]),
					thrust::raw_pointer_cast(&A[0]), nr_rows_A, nr_cols_A);
			break;
		case MUL:
			sMatMul(thrust::raw_pointer_cast(&C[0]), thrust::raw_pointer_cast(&A[0]),
					thrust::raw_pointer_cast(&B[0]), nr_rows_A, nr_cols_A, nr_cols_A);
			break;
		case SUM:
			sMatSum(thrust::raw_pointer_cast(&C[0]), thrust::raw_pointer_cast(&A[0]),
					thrust::raw_pointer_cast(&B[0]), nr_rows_A, nr_cols_A);
			break;
		case INV:

			break;
		default:
			break;
	}
	clock_t end = clock();
	printf("seq Mul took: %f seconds \n", float(end - start) / CLOCKS_PER_SEC);
	if(argc < 4 || atoi(argv[3]) != 0){
		print_matrix(thrust::raw_pointer_cast(&C[0]), nr_rows_A, nr_cols_A);
	}
	start = clock();
	switch (operation) {
		case TRANS:
			cublasMatTranspose(handle, thrust::raw_pointer_cast(&d_C[0]), thrust::raw_pointer_cast(&d_A[0]),
					thrust::raw_pointer_cast(&d_B[0]), nr_rows_A, nr_cols_A);
			break;
		case MUL:
			cublasMatMul(handle, thrust::raw_pointer_cast(&d_C[0]), thrust::raw_pointer_cast(&d_A[0]),
					thrust::raw_pointer_cast(&d_B[0]),	nr_rows_A, nr_cols_A, nr_cols_A);
			break;
		case SUM:
			cublasMatSum(handle, thrust::raw_pointer_cast(&d_C[0]), thrust::raw_pointer_cast(&d_A[0]),
					thrust::raw_pointer_cast(&d_B[0]), nr_rows_A, nr_cols_A);
			break;
		case INV:

			break;
		default:
			break;
	}

	end = clock();
	printf("par Mul took: %f seconds \n", float(end - start) / CLOCKS_PER_SEC);
	thrust::copy(d_C.begin(), d_C.end(), C.begin());
	// deallocate any capacity which may currently be associated with vec
	//vec.shrink_to_fit();

	if(argc < 4 || atoi(argv[3]) != 0){
		print_matrix(thrust::raw_pointer_cast(&C[0]), nr_rows_A, nr_cols_A);
	}

	start = clock();
	switch (operation) {
		case TRANS:
			pMatTranspose(thrust::raw_pointer_cast(&d_C[0]),
					thrust::raw_pointer_cast(&d_A[0]), nr_rows_A, nr_cols_A);
			break;
		case MUL:
			pMatMul(thrust::raw_pointer_cast(&d_C[0]), thrust::raw_pointer_cast(&d_A[0]),
					thrust::raw_pointer_cast(&d_B[0]),	nr_rows_A, nr_cols_A, nr_cols_A);
			break;
		case SUM:
			pMatSum(thrust::raw_pointer_cast(&d_C[0]), thrust::raw_pointer_cast(&d_A[0]),
					thrust::raw_pointer_cast(&d_B[0]), nr_rows_A, nr_cols_A);
			break;
		case INV:

			break;
		default:
			break;
	}

	end = clock();
	printf("My par Mul took: %f seconds \n", float(end - start) / CLOCKS_PER_SEC);
	thrust::copy(d_C.begin(), d_C.end(), C.begin());
	// deallocate any capacity which may currently be associated with vec
	//vec.shrink_to_fit();
	if(argc < 4 || atoi(argv[3]) != 0){
		print_matrix(thrust::raw_pointer_cast(&C[0]), nr_rows_A, nr_cols_A);
	}
	/*multComp(handle, A, B, C, d_A, d_B, d_C, nr_rows_A, nr_cols_A);
	sumComp(handle, A, B, C, d_A, d_B, d_C, nr_rows_A, nr_cols_A);*/
	hipblasDestroy(handle);
}
