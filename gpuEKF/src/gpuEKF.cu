#include "hip/hip_runtime.h"
/**
* Implement a parallel and a sequencial version of the EKF for SLAM.
*
* EKF(mean_t-1, covariance_t-1, control_t, observation_t)
* 
* 	mean'_t = g(control_t, mean_t-1)
* 	covariance'_t = Jacobian(g()) * covariance_t-1 * Jacobian(g())^T + OdometryError_t
* 
* 	KalmanGain = covariance'_t * Jacobian(h(observation_t)) * 
* 		(Jacobian(h(observation_t)) * covariance'_t * Jacobian(h(observation_t)) + ObservatioError_t)^-1
* 	mean_t = mean'_t + KalmanGain * (observation_t - h(observation_t))
* 	covariance_t = (I - KalmanGain* Jacobian(h(observation_t))) * covariance'_t
* 	return mean_t, covariance_t
 */
#include <stdio.h>
#include <stdlib.h>
#include <ctime>
#include <iostream>
#include<math.h>

#include<hipblas.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include"matrix.h"

#define TRANS 0
#define MUL 1
#define SUM 2
#define INV 3

//Important Index
#define MEAN_X (0*3 + 0)
#define MEAN_Y (1*3 + 1)
#define MEAN_TETA (2*3 + 2)
#define CONTROL_X 0 //Robot motion in the x axis
#define CONTROL_Y 1
#define CONTROL_TETA 2
#define FOCAL_LENGTH 0
#define K_X 1//is the number of pixels per unit length(k_x, k_y)
#define K_Y 2
#define CAM_X 3
#define CAM_Y 4
#define CAM_Z 5
#define L_EXISTS 6
#define LM_IND //landmark index related to the mean vector

void comp(int argc, char** argv){
	hipblasHandle_t handle;
	CUBLAS_CHECK_RETURN(hipblasCreate(&handle));
	int operation;
	// Allocate 3 arrays on CPU
	int nr_rows_A, nr_cols_A, nr_rows_B, nr_cols_B, nr_rows_C, nr_cols_C;

	if(argc < 3){
		nr_rows_A = nr_cols_A = nr_rows_B = nr_cols_B = nr_rows_C = nr_cols_C = 3;
		operation = 1;
	} else {
		nr_rows_A = nr_cols_A = nr_rows_B = nr_cols_B = nr_rows_C = nr_cols_C = atoi(argv[1]);
		operation = atoi(argv[2]);
	}

	thrust::device_vector<float> d_A(nr_rows_A * nr_cols_A),
			d_B(nr_rows_B * nr_cols_B), d_C(nr_rows_C * nr_cols_C);

	// Fill the arrays A and B on GPU with random numbers
	GPU_fill_rand(thrust::raw_pointer_cast(&d_A[0]), nr_rows_A, nr_cols_A);
	GPU_fill_rand(thrust::raw_pointer_cast(&d_B[0]), nr_rows_B, nr_cols_B);
	//thrust::device_vector<int> vec(d_A, d_A + n);

	thrust::host_vector<float>A(d_A.begin(), d_A.begin() + d_A.size());
	thrust::host_vector<float>B(d_B.begin(), d_B.begin() + d_B.size());
	thrust::host_vector<float>C(nr_rows_C * nr_cols_C);


	clock_t start = clock();
	switch (operation) {
		case TRANS:
			sMatTranspose(thrust::raw_pointer_cast(&C[0]),
					thrust::raw_pointer_cast(&A[0]), nr_rows_A, nr_cols_A);
			break;
		case MUL:
			sMatMul(NOT_TRANSP, NOT_TRANSP, thrust::raw_pointer_cast(&C[0]), thrust::raw_pointer_cast(&A[0]),
					thrust::raw_pointer_cast(&B[0]), nr_rows_A, nr_cols_A, nr_cols_A);
			break;
		case SUM:
			sMatSum(thrust::raw_pointer_cast(&C[0]), thrust::raw_pointer_cast(&A[0]),
					thrust::raw_pointer_cast(&B[0]), nr_rows_A, nr_cols_A);
			break;
		case INV:
			sMatInverse(thrust::raw_pointer_cast(&A[0]),
					nr_rows_A, nr_cols_A, thrust::raw_pointer_cast(&C[0]));
			break;
		default:
			break;
	}
	clock_t end = clock();
	printf("seq Mul took: %f seconds \n", float(end - start) / CLOCKS_PER_SEC);
	if(argc < 4 || atoi(argv[3]) != 0){
		print_matrix(thrust::raw_pointer_cast(&C[0]), nr_rows_A, nr_cols_A);
	}
	start = clock();
	switch (operation) {
		case TRANS:
			cublasMatTranspose(handle, thrust::raw_pointer_cast(&d_C[0]), thrust::raw_pointer_cast(&d_A[0]),
					thrust::raw_pointer_cast(&d_B[0]), nr_rows_A, nr_cols_A);
			break;
		case MUL:
			cublasMatMul(handle, NOT_TRANSP, NOT_TRANSP, thrust::raw_pointer_cast(&d_C[0]), thrust::raw_pointer_cast(&d_A[0]),
					thrust::raw_pointer_cast(&d_B[0]),	nr_rows_A, nr_cols_A, nr_cols_A);
			break;
		case SUM:
			cublasMatSum(handle, thrust::raw_pointer_cast(&d_C[0]), thrust::raw_pointer_cast(&d_A[0]),
					thrust::raw_pointer_cast(&d_B[0]), nr_rows_A, nr_cols_A);
			break;
		case INV:

			break;
		default:
			break;
	}

	end = clock();
	printf("par Mul took: %f seconds \n", float(end - start) / CLOCKS_PER_SEC);
	thrust::copy(d_C.begin(), d_C.end(), C.begin());
	// deallocate any capacity which may currently be associated with vec
	//vec.shrink_to_fit();

	if(argc < 4 || atoi(argv[3]) != 0){
		print_matrix(thrust::raw_pointer_cast(&C[0]), nr_rows_A, nr_cols_A);
	}

	start = clock();
	switch (operation) {
		case TRANS:
			pMatTranspose(thrust::raw_pointer_cast(&d_C[0]),
					thrust::raw_pointer_cast(&d_A[0]), nr_rows_A, nr_cols_A);
			break;
		case MUL:
			pMatMul(NOT_TRANSP, NOT_TRANSP, thrust::raw_pointer_cast(&d_C[0]), thrust::raw_pointer_cast(&d_A[0]),
					thrust::raw_pointer_cast(&d_B[0]),	nr_rows_A, nr_cols_A, nr_cols_A);
			break;
		case SUM:
			pMatSum(thrust::raw_pointer_cast(&d_C[0]), thrust::raw_pointer_cast(&d_A[0]),
					thrust::raw_pointer_cast(&d_B[0]), nr_rows_A, nr_cols_A);
			break;
		case INV:

			break;
		default:
			break;
	}

	end = clock();
	printf("My par Mul took: %f seconds \n", float(end - start) / CLOCKS_PER_SEC);
	thrust::copy(d_C.begin(), d_C.end(), C.begin());
	// deallocate any capacity which may currently be associated with vec
	//vec.shrink_to_fit();
	if(argc < 4 || atoi(argv[3]) != 0){
		print_matrix(thrust::raw_pointer_cast(&C[0]), nr_rows_A, nr_cols_A);
	}
	/*multComp(handle, A, B, C, d_A, d_B, d_C, nr_rows_A, nr_cols_A);
	sumComp(handle, A, B, C, d_A, d_B, d_C, nr_rows_A, nr_cols_A);*/
	hipblasDestroy(handle);
}

//TODO
float *odometryError(const float *control, int dim){
	float *error = (float *) calloc(dim * dim, sizeof(float));

	srand(time(NULL));
	error[0] = rand() * 0.2;
	error[dim + 1] = rand() * 0.2;
	error[2*dim + 2] = rand() * 0.2;

	return error;
}

float *moveUpadate(float *mean, const float *control){
	mean[MEAN_X] = mean[MEAN_X] + control[CONTROL_X] * cos(mean[MEAN_TETA] + control[CONTROL_TETA]/2);
	mean[MEAN_Y] = mean[MEAN_Y] + control[CONTROL_Y] * sin(mean[MEAN_TETA] + control[CONTROL_TETA]/2);
	mean[MEAN_TETA] = mean[MEAN_TETA] + control[CONTROL_TETA];
	
	return mean;
}

//TODO check if right
float *jacobianG(float teta, const float *control, int dim){
	float *G = (float *) calloc(dim * dim, sizeof(float));

	sCreateIdentity(G, dim);
	G[2] = sin(teta + control[CONTROL_TETA]/2);
	G[dim + 2] = cos(teta + control[CONTROL_TETA]/2);

	return G;
}

//TODO
float *observationError(const float *observation, int dim) {
	float *error = (float *) malloc(sizeof(float) * dim * dim);

	return error;
}

//TODO
float *jacobianH(float *observed, const float *observation, int dim){
	float *H = (float *) malloc(sizeof(float) * dim * dim);

	return H;
}

//TODO
float *getExpected(const float *observation, int dim){
	float *expected = (float *) calloc(3 * dim, sizeof(float));

	return expected;
}

//TODO
float *getObservation(const float *observation, int dim){
	float *observed = (float *) malloc(sizeof(float) * 3 * dim);

	return observed;
}

//TODO
bool landmarkExist(const float *observation) {
	if(observation[L_EXISTS] < 0)
		return false;
	return true;
}

//TODO
void addLandmark(float *mean, float *covariance, const float *observation, int *dim){
	*dim = *dim + 2;
	mean = (float *) realloc(mean, 3 * (*dim) * sizeof(float));
	covariance = (float *) realloc(covariance, sizeof(float) * (*dim) * (*dim));

	//Set the position(x,y) of the landmark in relation to the robot
	mean[(*dim - 1)*3] = mean[MEAN_X] + observation[FOCAL_LENGTH] *
			observation[K_X] * (observation[CAM_X]/observation[CAM_Z]);
	mean[(*dim - 1) * 3 + 1] = mean[MEAN_Y] + observation[FOCAL_LENGTH] *
			observation[K_Y] * (observation[CAM_Y]/observation[CAM_Z]);
}

void EKF(int dim, float *mean, float *covariance, const float *control, const float *observation){
	//******Update step******
	//u_t = g(control_t, mean_t-1)
	float teta = mean[MEAN_TETA];
	moveUpadate(mean, control);

	//TODO this block can be improved look the slides of stachness, and G is sparse
	float *partial = (float *)malloc(sizeof(float) * dim * dim);
	float *G = jacobianG(teta, control, dim);
	float *temp;
	//G * E_t-1
	sMatMul(NOT_TRANSP, NOT_TRANSP, partial, G, covariance, dim, dim, dim);
	//G * E_t-1 * G^T
	sMatMul(NOT_TRANSP, TRANSP, covariance, partial, G, dim, dim, dim);
	temp = odometryError(control, dim);
	//E = G * E_t-1 * G^T + OdometryError_t
	sMatSum(covariance, covariance, temp, dim, dim);
	free(G);
	free(temp);


	////******Matching and compute h(observation_t)******
	if(!landmarkExist(observation)){
		addLandmark(mean, covariance, observation, &dim);
		partial = (float *) realloc(partial, sizeof(float) * dim * dim);
	}
	float *expected = (float *) getExpected(observation, dim);
	

	//******Correction/Update step******
	//H is sparse, so this block can be improved...
	float *kalmanGain = (float *) malloc(sizeof(float) * dim * dim);
	float *H = jacobianH(observed, observation, dim);
	//E * H^T
	sMatMul(NOT_TRANSP, TRANSP, partial, H, covariance, dim, dim, dim);
	//H * E * H^T
	sMatMul(NOT_TRANSP, NOT_TRANSP, kalmanGain, H, partial, dim, dim, dim);
	//(H * E * H^T + observationError)
	temp = observationError(observation, dim);
	sMatSum(temp, kalmanGain, temp, dim, dim);
	//(H * E * H^T + observationError)^-1
	//check this, probabily wrong
	sMatInverse(temp, dim, dim, temp);
	//K = E * H^T * (H * E * H^T + observationError)^-1
	sMatMul(NOT_TRANSP, NOT_TRANSP, kalmanGain, partial, temp, dim, dim, dim);
	
	//z - h(u)
	float *expected = getExpected(observation, dim);
	sMatSub(observed, expected, observed, dim, 3);
	//K(z - h(u))
	sMatMul(NOT_TRANSP, NOT_TRANSP, expected, kalmanGain, observed, dim, dim, 3);
	free(observed);
	//u = u + K(z - h(u))
	sMatSum(mean, mean, expected, dim, 3);
	free(expected);

	//K * H
	sMatMul(NOT_TRANSP, NOT_TRANSP, partial, kalmanGain, H, dim, dim, dim);
	//(I - K * H)
	//temp = Identity matrix
	memset(temp, 0, sizeof(float));
	sCreateIdentity(temp, dim);
	//THIS can be improved...
	sMatSub(partial, temp, partial, dim, dim);
	free(kalmanGain);
	free(H);

	//E = (I - K * H) * E
	sMatMul(NOT_TRANSP, NOT_TRANSP, temp, partial, covariance, dim, dim, dim);
	free(covariance);
	covariance = temp;
	free(temp);
	free(partial);
}

//TODO
void parallelEKF(float *mean, float *covariance){
	
}

int main(int argc, char** argv) {
	comp(argc, argv);
}
