#include "hip/hip_runtime.h"
/**
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 */
#include <stdio.h>
#include <stdlib.h>
#include <ctime>
#include <iostream>

#include<hipblas.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include"matrix.h"
#define IDX2C (i ,j , ld ) ((( j )*( ld ))+( i ))
#define m 6 // a - mxk matrix
#define n 4 // b - kxn matrix
#define k 5 // c - mxn matrix


/**
 * Host function that prepares data array and passes it to the CUDA kernel.
 */
int main(int argc, char** argv) {
	hipblasHandle_t handle;
	CUBLAS_CHECK_RETURN(hipblasCreate(&handle));

	// Allocate 3 arrays on CPU
	int nr_rows_A, nr_cols_A, nr_rows_B, nr_cols_B, nr_rows_C, nr_cols_C;

	// for simplicity we are going to use square arrays
	nr_rows_A = nr_cols_A = nr_rows_B = nr_cols_B = nr_rows_C = nr_cols_C = 1000;

	thrust::device_vector<float> d_A(nr_rows_A * nr_cols_A),
			d_B(nr_rows_B * nr_cols_B), d_C(nr_rows_C * nr_cols_C);

	// Fill the arrays A and B on GPU with random numbers
	GPU_fill_rand(thrust::raw_pointer_cast(&d_A[0]), nr_rows_A, nr_cols_A);
	GPU_fill_rand(thrust::raw_pointer_cast(&d_B[0]), nr_rows_B, nr_cols_B);
	//thrust::device_vector<int> vec(d_A, d_A + n);

	thrust::host_vector<float>A(d_A.begin(), d_A.begin() + d_A.size());
	thrust::host_vector<float>B(d_B.begin(), d_B.begin() + d_B.size());
	thrust::host_vector<float>C(nr_rows_C * nr_cols_C);

	clock_t start = clock();
	sMatMul(thrust::raw_pointer_cast(&C[0]), thrust::raw_pointer_cast(&A[0]),
			thrust::raw_pointer_cast(&B[0]), nr_rows_A, nr_cols_A, nr_cols_B);
	clock_t end = clock();
	printf("seq Mul took: %f seconds \n", float(end - start) / CLOCKS_PER_SEC);
	//print_matrix(thrust::raw_pointer_cast(&C[0]), nr_rows_C, nr_cols_C);

	start = clock();
	cublasMatMul(handle, thrust::raw_pointer_cast(&d_C[0]), thrust::raw_pointer_cast(&d_A[0]),
			thrust::raw_pointer_cast(&d_B[0]),	nr_rows_A, nr_cols_A, nr_cols_B);
	end = clock();

	printf("seq Mul took: %f seconds \n", float(end - start) / CLOCKS_PER_SEC);
	//thrust::copy(d_C.begin(), d_C.end(), C.begin());
	//print_matrix(thrust::raw_pointer_cast(&C[0]), nr_rows_C, nr_cols_C);
	hipblasDestroy(handle);
}
