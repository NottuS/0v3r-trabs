#include "hip/hip_runtime.h"
#include "matrix.h"

__global__ void kernelCreateIdentity(float *I, int nr_rows_I){
	int id = blockIdx.x * blockDim.x + threadIdx.x;

	if (id < nr_cols_A) {
		I[id * nr_rows_I + id] = 1;
	}
}

void createIdentity(float *I, int nr_rows_I){
	dim3 dimBlock(BLOCK_START_SIZE);
	dim3 dimGrid(ceil(nr_rows_A/dimBlock));

	kernelCreateIdentity<<<dimGrid,dimBlock>>>(I, nr_rows_I);
}

void choleskyDecomp(float *A, float *L, int nr_rows_A, int nr_cols_A){
	int i,j,k;
	for(i = 0; i < nr_rows_A; i++) {
		for (j = 0; j < i; ++j) {
			if(i == j) {
				for(j = 1; i - 2; j++){
						sum += L[i][j] << 1;
				}
				L[i][i] = sqrt(A[i][i] - sum);
				sum = 0;
			} else {
				for(j = 0; k - 2; j++){
					sum += L[k][j] * L[i][j];
				}

				L[i][k] = (A[i][k] - sum) / L[k][k];
				sum = 0;
			}
		}
	}
}

void cublasMatInverse(hipblasHandle_t &handle, const float *A, int nr_rows_A, int nr_cols_A, float *result){

	const float alf = 1;
	const float *alpha = &alf;

	//create the identity matrix
	thrust::device_vector<float> I(nr_rows_A * nr_cols_A);
	thrust::host_vector<float> L(nr_rows_A * nr_cols_A);
	thust::fill(I.begin(), I.size(), 0);
	hipDeviceSynchronize();
	createIdentity(thrust::raw_pointer_cast(&I[0]), nr_cols_A);
	hipDeviceSynchronize();

	//get L from A = LL^T
	choleskyDecomp(A,L, nr_rows_A, nr_cols_A);
	thrust::device_vector<float> d_L(L.begin, L.begin() + L.size());

	//get Y from LY = I
	hipblasStrsm(handle,HIPBLAS_SIDE_RIGHT, HIPBLAS_FILL_MODE_LOWER,
			HIPBLAS_OP_N,HIPBLAS_DIAG_NON_UNIT, nr_rows_A, nr_cols_A, alf, A, nr_rows_A, I, nr_rows_A);
	hipDeviceSynchronize();

	//get X from (L^T)X = Y
	//TODO

	d_L.shrink_to_fit();
	L.shrink_to_fit();
	result = thrust::raw_pointer_cast(&I[0]);
}
