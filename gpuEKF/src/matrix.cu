#include"matrix.h"
#include <stdio.h>

void sMatMul(float *C, const float *A, const float *B,
		unsigned int hA, unsigned int wA, unsigned int wB)
{
	for (unsigned int i = 0; i < hA; ++i)
		for (unsigned int j = 0; j < wB; ++j) {
			double sum = 0;
			for (unsigned int k = 0; k < wA; ++k) {
				double a = A[i * wA + k];
				double b = B[k * wB + j];
				sum += a * b;
			}
			C[i * wB + j] = (float)sum;
		}
}

void pMatMul(float *C, const float *A, const float *B, unsigned int hA, unsigned int wA, unsigned int wB){

}

void cublasMatMul(hipblasHandle_t &handle, float *C,
		const float *A, const float *B, unsigned int m, unsigned int k, unsigned int n){
	const float alf = 1;
	const float bet = 0;
	const float *alpha = &alf;
	const float *beta = &bet;

	// Do the actual multiplication
	// matrix - matrix multiplication : C = alf*A*B + bet*C
	// A -mxk matrix , B -kxn matrix , C -mxn matrix ;
	CUBLAS_CHECK_RETURN(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
			m, n, k, alpha, A, m, B, k, beta, C, m));
}
