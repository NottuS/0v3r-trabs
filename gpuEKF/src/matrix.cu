#include "hip/hip_runtime.h"
#include"matrix.h"
#include <stdio.h>

//Print matrix A(nr_rows_A, nr_cols_A) storage in column-major format
void print_matrix(const float *A, int nr_rows_A, int nr_cols_A) {

    for(int i = 0; i < nr_rows_A; ++i){
        for(int j = 0; j < nr_cols_A; ++j){
            std::cout << A[j * nr_rows_A + i] << " ";
        }
        std::cout << std::endl;
    }
    std::cout << std::endl;
}

__global__ void kernelCreateIdentity(float *I, int nr_rows_I){
	int id = blockIdx.x * blockDim.x + threadIdx.x;

	if (id < nr_rows_I) {
		I[id * nr_rows_I + id] = 1;
	}
}

void createIdentity(float *I, int nr_rows_I){
	dim3 dimBlock(BLOCK_START_SIZE);
	dim3 dimGrid(ceil(float(nr_rows_I)/dimBlock.x));

	kernelCreateIdentity<<<dimGrid,dimBlock>>>(I, nr_rows_I);
}

void sMatMul(float *C, const float *A, const float *B,
		unsigned int hA, unsigned int wA, unsigned int wB)
{
	for (unsigned int i = 0; i < hA; ++i)
		for (unsigned int j = 0; j < wB; ++j) {
			double sum = 0;
			for (unsigned int k = 0; k < wA; ++k) {
				double a = A[i * wA + k];
				double b = B[k * wB + j];
				sum += a * b;
			}
			C[i * wB + j] = (float)sum;
		}
}

__global__ void kernelMatMul(float *C, const float *A, const float *B, unsigned int nr_rows_A, unsigned int nr_cols_A, unsigned int nr_cols_B){
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;


	__shared__ float tempA[BLOCK_START_SIZE];
	__shared__ float tempB[BLOCK_START_SIZE];

	float Cvalue = 0;
	int stride = 1;

	if (row < nr_rows_A && col < nr_cols_B) {
		//TODO Check if all ther work is done, and the indexs
		for (int i = 0; i < nr_rows_A/(BLOCK_START_SIZE); ++i)
		{
			tempA[threadIdx.y*BLOCK_SIZE + threadIdx.x] = A[row * nr_col_A + i * BLOCK_START_SIZE + threadIdx.x];
			tempB[threadIdx.y*BLOCK_SIZE + threadIdx.x] = B[row * nr_rows_A + i * BLOCK_START_SIZE + threadIdx.x];

			__syncthreads();
			for (int j = 0; j < BLOCK_START_SIZE; ++j)
				Cvalue += tempB[j] * tempA[j];
			stride++;
			__syncthreads();
		}
		C[row * nr_cols_A + col] = Cvalue;
		
	}
}

void pMatMul(float *C, const float *A, const float *B, unsigned int hA, unsigned int wA, unsigned int wB){
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid(ceil(float(nr_rows_A) / dimBlock.x), ceil(float(nr_cols_B) / dimBlock.y));

	//hipSetDeviceFlags(hipDeviceLmemResizeToMax);
	kernelMatMul<<<dimGrid, dimBlock>>>(C,A,B,nr_rows_A, nr_cols_A);
}

void cublasMatMul(hipblasHandle_t &handle, float *C,
		const float *A, const float *B, unsigned int m, unsigned int k, unsigned int n){
	const float alf = 1;
	const float bet = 0;
	const float *alpha = &alf;
	const float *beta = &bet;

	// Do the actual multiplication
	// matrix - matrix multiplication : C = alf*A*B + bet*C
	// A -mxk matrix , B -kxn matrix , C -mxn matrix ;
	CUBLAS_CHECK_RETURN(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
			m, n, k, alpha, A, m, B, k, beta, C, m));
}

void sMatSum(float *C, const float *A, const float *B, int nr_rows_A, int nr_cols_A){
	for(int i = 0; i < nr_rows_A; i++){
		for (int j = 0; j < nr_cols_A; ++j) {
			C[i*nr_cols_A + j] = A[i*nr_cols_A + j] + B[i*nr_cols_A + j];
		}
	}
}

__global__ void kernelMatSum(float *C, const float *A, const float *B, int nr_rows_A, int nr_cols_A){
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	if (row < nr_rows_A && col < nr_cols_A) {
		C[row*nr_cols_A + col] = A[row*nr_cols_A + col] + B[row*nr_cols_A + col];
	}
}

void pMatSum(float *C, const float *A, const float *B, int nr_rows_A, int nr_cols_A){
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid(ceil(float(nr_rows_A) / dimBlock.x), ceil(float(nr_cols_A) / dimBlock.y));

	//hipSetDeviceFlags(hipDeviceLmemResizeToMax);
	kernelMatSum<<<dimGrid, dimBlock>>>(C,A,B,nr_rows_A, nr_cols_A);
	//hipDeviceReset();
}

void cublasMatSum(hipblasHandle_t &handle, float *C, const float *A, const float *B, int nr_rows_A, int nr_cols_A){
	const float alf = 1;
	const float bet = 1;
	const float *alpha = &alf;
	const float *beta = &bet;

	CUBLAS_CHECK_RETURN(hipblasSgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, nr_rows_A, nr_cols_A, alpha,
			A, nr_rows_A, beta, B, nr_rows_A, C, nr_rows_A));
}

void sMatSub(float *C, const float *A, const float *B, int nr_rows_A, int nr_cols_A){
	for(int i = 0; i < nr_rows_A; i++){
		for (int j = 0; j < nr_cols_A; ++j) {
			C[i*nr_cols_A + j] = A[i*nr_cols_A + j] - B[i*nr_cols_A + j];
		}
	}
}

__global__ void kernelMatSub(float *C, const float *A, const float *B, int nr_rows_A, int nr_cols_A){
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	if (row < nr_rows_A && col < nr_cols_A) {
		C[row*nr_cols_A + col] = A[row*nr_cols_A + col] - B[row*nr_cols_A + col];
	}
}

void pMatSub(float *C, const float *A, const float *B, int nr_rows_A, int nr_cols_A){
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid(ceil(float(nr_rows_A) / dimBlock.x), ceil(float(nr_cols_A) / dimBlock.y));


	//hipSetDeviceFlags(hipDeviceLmemResizeToMax);
	kernelMatSub<<<dimGrid, dimBlock>>>(C,A,B,nr_rows_A, nr_cols_A);
	//hipDeviceReset();
}

void cublasMatSub(hipblasHandle_t &handle, float *C, const float *A, const float *B, int nr_rows_A, int nr_cols_A){
	const float alf = 1;
	const float bet = -1;
	const float *alpha = &alf;
	const float *beta = &bet;

	CUBLAS_CHECK_RETURN(hipblasSgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, nr_rows_A, nr_cols_A, alpha,
			A, nr_rows_A, beta, B, nr_rows_A, C, nr_rows_A));
}

void sMatTranspose(float *C, const float *A, int nr_rows_A, int nr_cols_A){
	for (int i = 0; i < nr_cols_A; ++i) {
		for (int j = 0; j < nr_rows_A; ++j) {
			C[i * nr_cols_A + j] = A[j * nr_rows_A + i];
		}
	}
}

__global__ void kernelTranspose(float *C, const float *A, int nr_rows_A, int nr_cols_A){
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	__shared__ float temp[BLOCK_SIZE][BLOCK_SIZE+1];

	if (row < nr_rows_A && col < nr_cols_A) {
		int id_in = row * nr_cols_A + col;
		temp[threadIdx.y][threadIdx.x] = A[id_in];
	}

	__syncthreads();

	col = blockIdx.y * blockDim.y + threadIdx.y;
	row = blockIdx.x * blockDim.x + threadIdx.x;

	if (row < nr_cols_A && col < nr_rows_A) {
		int id_out = col * nr_rows_A + row;
		C[id_out] = temp[threadIdx.x][threadIdx.y];
	}
}

void pMatTranspose(float *C, const float *A, int nr_rows_A, int nr_cols_A){
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid(ceil(float(nr_rows_A) / dimBlock.x), ceil(float(nr_cols_A) / dimBlock.y));

	kernelTranspose<<<dimGrid, dimBlock>>>(C, A, nr_rows_A, nr_cols_A);
}

void cublasMatTranspose(hipblasHandle_t &handle, float *C, const float *A, const float *B, int nr_rows_A, int nr_cols_A){
	const float alf = 1;
	const float bet = 0;
	const float *alpha = &alf;
	const float *beta = &bet;

	CUBLAS_CHECK_RETURN(hipblasSgeam(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, nr_rows_A, nr_cols_A, alpha,
			A, nr_rows_A, beta, B, nr_rows_A, C, nr_rows_A));
}

void sInvMat(const float *A, int nr_rows_A, int nr_cols_A){

}
void pInvMat(const float *A, int nr_rows_A, int nr_cols_A){

}

void choleskyDecomp(const float *A, float *L, int nr_rows_A, int nr_cols_A){
	int i,j,k;
	float sum;
	for(i = 0; i < nr_rows_A; i++) {
		for (j = 0; j < i; ++j) {
			if(i == j) {
				for(j = 1; i - 2; j++){
						sum += L[i*nr_cols_A + j] * L[i*nr_cols_A + j];
				}
				L[i*nr_cols_A + i] = sqrt(A[i*nr_cols_A + i] - sum);
				sum = 0;
			} else {
				for(j = 0; k - 2; j++){
					sum += L[k*nr_cols_A +j] * L[i*nr_cols_A + j];
				}
				L[i*nr_cols_A + k] = (A[i*nr_cols_A + k] - sum) / L[k*nr_cols_A + k];
				sum = 0;
			}
		}
	}
}

void cublasMatInverse(hipblasHandle_t &handle, const float *A, int nr_rows_A, int nr_cols_A, float *result){

	const float alf = 1;
	const float *alpha = &alf;

	//create the identity matrix
	thrust::device_vector<float> I(nr_rows_A * nr_cols_A);
	thrust::host_vector<float> L(nr_rows_A * nr_cols_A);
	thrust::fill(I.begin(), I.end(), 0);
	hipDeviceSynchronize();
	createIdentity(thrust::raw_pointer_cast(&I[0]), nr_cols_A);
	hipDeviceSynchronize();

	//get L from A = LL^T
	choleskyDecomp(A, thrust::raw_pointer_cast(&L[0]), nr_rows_A, nr_cols_A);
	thrust::device_vector<float> d_L(L.begin(), L.begin() + L.size());

	//get Y from LY = I
	hipblasStrsm(handle,HIPBLAS_SIDE_RIGHT, HIPBLAS_FILL_MODE_LOWER,
			HIPBLAS_OP_N,HIPBLAS_DIAG_NON_UNIT, nr_rows_A, nr_cols_A, alpha, A,
			nr_rows_A, thrust::raw_pointer_cast(&I[0]), nr_rows_A);
	hipDeviceSynchronize();

	//get X from (L^T)X = Y
	hipblasStrsm(handle, HIPBLAS_SIDE_RIGHT, HIPBLAS_FILL_MODE_UPPER,
				HIPBLAS_OP_T,HIPBLAS_DIAG_NON_UNIT, nr_rows_A, nr_cols_A,	alpha,
				thrust::raw_pointer_cast(&L[0]), nr_rows_A, thrust::raw_pointer_cast(&I[0]), nr_rows_A);
	hipDeviceSynchronize();
	d_L.shrink_to_fit();
	L.shrink_to_fit();
	result = thrust::raw_pointer_cast(&I[0]);
}


