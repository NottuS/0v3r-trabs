#include"matrix.h"
#include <stdio.h>

void sMatMul(float *C, const float *A, const float *B,
		unsigned int hA, unsigned int wA, unsigned int wB)
{
	for (unsigned int i = 0; i < hA; ++i)
		for (unsigned int j = 0; j < wB; ++j) {
			double sum = 0;
			for (unsigned int k = 0; k < wA; ++k) {
				double a = A[i * wA + k];
				double b = B[k * wB + j];
				sum += a * b;
			}
			C[i * wB + j] = (float)sum;
		}
}

void pMatMul(float *C, const float *A, const float *B, unsigned int hA, unsigned int wA, unsigned int wB){

}

void cublasMatMul(hipblasHandle_t &handle, float *C,
		const float *A, const float *B, unsigned int m, unsigned int k, unsigned int n){
	const float alf = 1;
	const float bet = 0;
	const float *alpha = &alf;
	const float *beta = &bet;

	// Do the actual multiplication
	// matrix - matrix multiplication : C = alf*A*B + bet*C
	// A -mxk matrix , B -kxn matrix , C -mxn matrix ;
	CUBLAS_CHECK_RETURN(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
			m, n, k, alpha, A, m, B, k, beta, C, m));
}

//Print matrix A(nr_rows_A, nr_cols_A) storage in column-major format
void print_matrix(const float *A, int nr_rows_A, int nr_cols_A) {

    for(int i = 0; i < nr_rows_A; ++i){
        for(int j = 0; j < nr_cols_A; ++j){
            std::cout << A[j * nr_rows_A + i] << " ";
        }
        std::cout << std::endl;
    }
    std::cout << std::endl;
}
