#include "hip/hip_runtime.h"
#include"matrix.h"
#include <stdio.h>

//using namespace Eigen;

//Print matrix A(nr_rows_A, nr_cols_A) storage in column-major format
void print_matrix(const float *A, int nr_rows_A, int nr_cols_A) {

    for(int i = 0; i < nr_rows_A; ++i){
        for(int j = 0; j < nr_cols_A; ++j){
            std::cout << A[i * nr_rows_A + j] << " ";
        }
        std::cout << std::endl;
    }
    std::cout << std::endl;
}

__global__ void kernelCreateIdentity(float *I, int nr_rows_I){
	int id = blockIdx.x * blockDim.x + threadIdx.x;

	if (id < nr_rows_I) {
		I[id * nr_rows_I + id] = 1;
	}
}

void createIdentity(float *I, int nr_rows_I){
	dim3 dimBlock(BLOCK_START_SIZE);
	dim3 dimGrid(ceil(float(nr_rows_I)/dimBlock.x));

	kernelCreateIdentity<<<dimGrid,dimBlock>>>(I, nr_rows_I);
}

void sCreateIdentity(float *I, int nr_rows_I){
	for(int i = 0; i < nr_rows_I; i++){
		I[i*nr_rows_I + i] = 1;
	}
}

//TODO allow transpose on multiplication
void sMatMul(int transp_1, int transp_2,float *C, const float *A, const float *B,
		unsigned int hA, unsigned int wA, unsigned int wB)
{
	for (unsigned int i = 0; i < hA; ++i)
		for (unsigned int j = 0; j < wB; ++j) {
			double sum = 0;
			for (unsigned int k = 0; k < wA; ++k) {
				double a;
				double b;
				if(! transp_1)
					a = A[i * wA + k];
				else
					a = A[k * wA + i];
				if(! transp_2)
					b = B[k * wB + j];
				else
					b = B[j * wB + k];
				sum += a * b;
			}
			C[i * wB + j] = (float)sum;
		}
}

__global__ void kernelMatMul(float *C, const float *A,
		const float *B, unsigned int nr_rows_A, unsigned int nr_cols_A, unsigned int nr_cols_B){
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	__shared__ float tempA[BLOCK_SIZE][BLOCK_SIZE];
	__shared__ float tempB[BLOCK_SIZE][BLOCK_SIZE];

	float Cvalue = 0;

	if (row < nr_rows_A && col < nr_cols_B) {
		//TODO Check if all their work is done, and the indexs
		for (int i = 0; i < ceil(float(nr_rows_A)/(BLOCK_SIZE)); ++i)
		{
			/*tempA[threadIdx.y*BLOCK_SIZE + threadIdx.x] = A[row * nr_col_A + i * BLOCK_START_SIZE + threadIdx.x];
			tempB[threadIdx.y*BLOCK_SIZE + threadIdx.x] = B[(i * BLOCK_START_SIZE + threadIdx.x)* nr_cols_B *  + row];*/
			tempA[threadIdx.y][threadIdx.x] = A[threadIdx.y * nr_cols_A + BLOCK_SIZE * i + threadIdx.x];
			tempB[threadIdx.y][threadIdx.x] = B[(BLOCK_SIZE * i + threadIdx.y) * nr_cols_B + threadIdx.x];
			__syncthreads();

			#pragma unroll
			for (int j = 0; j < BLOCK_SIZE; ++j)
				Cvalue += 1*tempA[threadIdx.y][j] * tempB[j][threadIdx.x] + 0*C[row * nr_cols_A + col] ;

			__syncthreads();
		}
		C[row * nr_cols_A + col] = Cvalue;
		
	}
}

__global__ void transp1kernelMatMul(float *C, const float *A,
		const float *B, unsigned int nr_rows_A, unsigned int nr_cols_A, unsigned int nr_cols_B){
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	__shared__ float tempA[BLOCK_SIZE][BLOCK_SIZE];
	__shared__ float tempB[BLOCK_SIZE][BLOCK_SIZE];

	float Cvalue = 0;

	if (row < nr_rows_A && col < nr_cols_B) {
		//TODO Check if all their work is done, and the indexs
		for (int i = 0; i < ceil(float(nr_rows_A)/(BLOCK_SIZE)); ++i)
		{
			/*tempA[threadIdx.y*BLOCK_SIZE + threadIdx.x] = A[row * nr_col_A + i * BLOCK_START_SIZE + threadIdx.x];
			tempB[threadIdx.y*BLOCK_SIZE + threadIdx.x] = B[(i * BLOCK_START_SIZE + threadIdx.x)* nr_cols_B *  + row];*/
			tempA[threadIdx.y][threadIdx.x] = A[(BLOCK_SIZE * i + threadIdx.y) * nr_cols_A + threadIdx.x];
			tempB[threadIdx.y][threadIdx.x] = B[(BLOCK_SIZE * i + threadIdx.y) * nr_cols_B + threadIdx.x];
			__syncthreads();

			#pragma unroll
			for (int j = 0; j < BLOCK_SIZE; ++j)
				Cvalue += 1*tempA[j][threadIdx.y] * tempB[j][threadIdx.x] + 0*C[row * nr_cols_A + col] ;

			__syncthreads();
		}
		C[row * nr_cols_A + col] = Cvalue;

	}
}

__global__ void transp2kernelMatMul(float *C, const float *A,
		const float *B, unsigned int nr_rows_A, unsigned int nr_cols_A, unsigned int nr_cols_B){
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	__shared__ float tempA[BLOCK_SIZE][BLOCK_SIZE];
	__shared__ float tempB[BLOCK_SIZE][BLOCK_SIZE];

	float Cvalue = 0;

	if (row < nr_rows_A && col < nr_cols_B) {
		//TODO Check if all their work is done, and the indexs
		for (int i = 0; i < ceil(float(nr_rows_A)/(BLOCK_SIZE)); ++i)
		{
			/*tempA[threadIdx.y*BLOCK_SIZE + threadIdx.x] = A[row * nr_col_A + i * BLOCK_START_SIZE + threadIdx.x];
			tempB[threadIdx.y*BLOCK_SIZE + threadIdx.x] = B[(i * BLOCK_START_SIZE + threadIdx.x)* nr_cols_B *  + row];*/
			tempA[threadIdx.y][threadIdx.x] = A[threadIdx.y * nr_cols_A + BLOCK_SIZE * i + threadIdx.x];
			tempB[threadIdx.y][threadIdx.x] = B[threadIdx.y * nr_cols_B + BLOCK_SIZE * i + threadIdx.x];
			__syncthreads();

			#pragma unroll
			for (int j = 0; j < BLOCK_SIZE; ++j)
				Cvalue += 1*tempA[threadIdx.y][j] * tempB[threadIdx.x][j] + 0*C[row * nr_cols_A + col] ;

			__syncthreads();
		}
		C[row * nr_cols_A + col] = Cvalue;

	}
}

//TODO allow transpose on multiplication
void pMatMul(int transp_1, int transp_2, float *C, const float *A, const float *B,
		unsigned int nr_rows_A, unsigned int nr_cols_A, unsigned int nr_cols_B){
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid(ceil(float(nr_rows_A) / dimBlock.x), ceil(float(nr_cols_B) / dimBlock.y));

	//hipSetDeviceFlags(hipDeviceLmemResizeToMax);
	if(transp_1)
		transp1kernelMatMul<<<dimGrid, dimBlock>>>(C, A, B, nr_rows_A, nr_cols_A, nr_cols_B);
	else if(transp_2)
		transp2kernelMatMul<<<dimGrid, dimBlock>>>(C, A, B, nr_rows_A, nr_cols_A, nr_cols_B);
	else
		kernelMatMul<<<dimGrid, dimBlock>>>(C, A, B, nr_rows_A, nr_cols_A, nr_cols_B);
}

void cublasMatMul(hipblasHandle_t &handle, int transp_1, int transp_2, float *C,
		const float *A, const float *B, unsigned int m, unsigned int k, unsigned int n){
	const float alf = 1;
	const float bet = 0;
	const float *alpha = &alf;
	const float *beta = &bet;

	// Do the actual multiplication
	// matrix - matrix multiplication : C = alf*A*B + bet*C
	// A -mxk matrix , B -kxn matrix , C -mxn matrix ;
	CUBLAS_CHECK_RETURN(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
			m, n, k, alpha, A, m, B, k, beta, C, m));
}

void sMatSum(float *C, const float *A, const float *B, int nr_rows_A, int nr_cols_A){
	for(int i = 0; i < nr_rows_A * nr_cols_A; i++){
		C[i] = A[i] + B[i];
	}

}

__global__ void kernelMatSum(float *C, const float *A, const float *B, int nr_rows_A, int nr_cols_A){
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	if (row < nr_rows_A && col < nr_cols_A) {
		C[row*nr_cols_A + col] = A[row*nr_cols_A + col] + B[row*nr_cols_A + col];
	}
}

void pMatSum(float *C, const float *A, const float *B, int nr_rows_A, int nr_cols_A){
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid(ceil(float(nr_rows_A) / dimBlock.x), ceil(float(nr_cols_A) / dimBlock.y));

	//hipSetDeviceFlags(hipDeviceLmemResizeToMax);
	kernelMatSum<<<dimGrid, dimBlock>>>(C,A,B,nr_rows_A, nr_cols_A);
	//hipDeviceReset();
}

void cublasMatSum(hipblasHandle_t &handle, float *C, const float *A, const float *B, int nr_rows_A, int nr_cols_A){
	const float alf = 1;
	const float bet = 1;
	const float *alpha = &alf;
	const float *beta = &bet;

	CUBLAS_CHECK_RETURN(hipblasSgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, nr_rows_A, nr_cols_A, alpha,
			A, nr_rows_A, beta, B, nr_rows_A, C, nr_rows_A));
}

void sMatSub(float *C, const float *A, const float *B, int nr_rows_A, int nr_cols_A){
	for(int i = 0; i < nr_rows_A * nr_cols_A; i++){
		C[i] = A[i] - B[i];
	}
}

__global__ void kernelMatSub(float *C, const float *A, const float *B, int nr_rows_A, int nr_cols_A){
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	if (row < nr_rows_A && col < nr_cols_A) {
		C[row*nr_cols_A + col] = A[row*nr_cols_A + col] - B[row*nr_cols_A + col];
	}
}

void pMatSub(float *C, const float *A, const float *B, int nr_rows_A, int nr_cols_A){
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid(ceil(float(nr_rows_A) / dimBlock.x), ceil(float(nr_cols_A) / dimBlock.y));


	//hipSetDeviceFlags(hipDeviceLmemResizeToMax);
	kernelMatSub<<<dimGrid, dimBlock>>>(C,A,B,nr_rows_A, nr_cols_A);
	//hipDeviceReset();
}

void cublasMatSub(hipblasHandle_t &handle, float *C, const float *A, const float *B, int nr_rows_A, int nr_cols_A){
	const float alf = 1;
	const float bet = -1;
	const float *alpha = &alf;
	const float *beta = &bet;

	CUBLAS_CHECK_RETURN(hipblasSgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, nr_rows_A, nr_cols_A, alpha,
			A, nr_rows_A, beta, B, nr_rows_A, C, nr_rows_A));
}

void sMatTranspose(float *C, const float *A, int nr_rows_A, int nr_cols_A){
	for (int i = 0; i < nr_cols_A; ++i) {
		for (int j = 0; j < nr_rows_A; ++j) {
			C[i * nr_cols_A + j] = A[j * nr_rows_A + i];
		}
	}
}

__global__ void kernelTranspose(float *C, const float *A, int nr_rows_A, int nr_cols_A){
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	__shared__ float temp[BLOCK_SIZE][BLOCK_SIZE+1];

	if (row < nr_rows_A && col < nr_cols_A) {
		int id_in = row * nr_cols_A + col;
		temp[threadIdx.y][threadIdx.x] = A[id_in];
	}

	__syncthreads();

	col = blockIdx.y * blockDim.y + threadIdx.y;
	row = blockIdx.x * blockDim.x + threadIdx.x;

	if (row < nr_cols_A && col < nr_rows_A) {
		int id_out = col * nr_rows_A + row;
		C[id_out] = temp[threadIdx.x][threadIdx.y];
	}
}

void pMatTranspose(float *C, const float *A, int nr_rows_A, int nr_cols_A){
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid(ceil(float(nr_rows_A) / dimBlock.x), ceil(float(nr_cols_A) / dimBlock.y));

	kernelTranspose<<<dimGrid, dimBlock>>>(C, A, nr_rows_A, nr_cols_A);
}

void cublasMatTranspose(hipblasHandle_t &handle, float *C, const float *A, const float *B, int nr_rows_A, int nr_cols_A){
	const float alf = 1;
	const float bet = 0;
	const float *alpha = &alf;
	const float *beta = &bet;

	CUBLAS_CHECK_RETURN(hipblasSgeam(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, nr_rows_A, nr_cols_A, alpha,
			A, nr_rows_A, beta, B, nr_rows_A, C, nr_rows_A));
}



void sMatInverse(float *A, int nr_rows_A, int nr_cols_A, float *resultado){
	/*thrust::device_vector<float> I(nr_rows_A * nr_cols_A);
	thrust::host_vector<float> L(nr_rows_A * nr_cols_A);
	thrust::fill(I.begin(), I.end(), 0);
	hipDeviceSynchronize();
	createIdentity(thrust::raw_pointer_cast(&I[0]), nr_rows_A);
	hipDeviceSynchronize();*/

	//get L from A = LL^T
	/*choleskyDecomp(A, thrust::raw_pointer_cast(&L[0]), nr_rows_A, nr_cols_A);
*/
	gsl_matrix_float_view gA = gsl_matrix_float_view_array(A, nr_rows_A, nr_cols_A);
	//gsl_matrix_float *gA = gsl_matrix_float_alloc (nr_rows_A, nr_cols_A);
	gsl_linalg_float_cholesky_decomp(&gA.matrix);
	gsl_linalg_float_cholesky_invert(&gA.matrix);
}

void choleskyDecomp(const float *A, float *L, int nr_rows_A, int nr_cols_A){
	int i,j,k;
	float sum;
	for(i = 0; i < nr_rows_A; i++) {
		for (j = 0; j <= i; ++j) {
			sum = 0;
			for(k = 0; k < j; k++) {
				sum += L[i*nr_cols_A +k] * L[j*nr_cols_A + k];
			}
			sum -= A[i*nr_cols_A + j];
			if(i == j) {
				L[i*nr_cols_A + j] = sqrt(sum);
			} else {
				L[i*nr_cols_A + j] = sum / L[j*nr_cols_A + j];
			}
		}
	}
}

__global__ void choleskyDecompKernel(int ind, const float *A, float *L, float diagonal, int nr_rows_A, int nr_cols_A){
	/*int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;*/
	int row = blockIdx.x * blockDim.x + threadIdx.x;
	__shared__ float temp[BLOCK_START_SIZE];
	
	if (row < nr_rows_A) {
		int x = threadIdx.x, y = threadIdx.y, problem = blockIdx.x;
		//temp[x] = -A[row + ind];
		for(int i = 0; i < ind; i++){
			L[row + ind] += L[row + i] * L[ ind * nr_cols_A + i];
		}
		//run Gauss-Jordan in shared memory (see next slide)
		#pragma unroll
		for( int i = 0; i < BLOCK_SIZE; i++ ) {
			if( y == i ) temp[y][x] /= temp[i][i];
			__syncthreads( );
			if( y != i ) temp[y][x] -= temp[y][i]*temp[i][x];
		}
		//copy result to global memory
		L[32*32*problem+32*y+x] = temp[y][x];
	}
}


//TODO
void pMatInverse(const float *A, int nr_rows_A, int nr_cols_A, float *result){
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid(ceil(float(nr_rows_A) / dimBlock.x), ceil(float(nr_cols_A) / dimBlock.y));
}

//TODO correct this one
void cublasMatInverse(hipblasHandle_t &handle, const float *A, int nr_rows_A, int nr_cols_A, float *result){

	const float alf = 1;
	const float *alpha = &alf;

	//create the identity matrix
	thrust::device_vector<float> I(nr_rows_A * nr_cols_A);
	thrust::host_vector<float> L(nr_rows_A * nr_cols_A);
	thrust::fill(I.begin(), I.end(), 0);
	hipDeviceSynchronize();
	createIdentity(thrust::raw_pointer_cast(&I[0]), nr_rows_A);
	hipDeviceSynchronize();

	//get L from A = LL^T
	choleskyDecomp(A, thrust::raw_pointer_cast(&L[0]), nr_rows_A, nr_cols_A);
	thrust::device_vector<float> d_L(L.begin(), L.begin() + L.size());

	//get Y from LY = I
	hipblasStrsm(handle,HIPBLAS_SIDE_RIGHT, HIPBLAS_FILL_MODE_LOWER,
			HIPBLAS_OP_N,HIPBLAS_DIAG_NON_UNIT, nr_rows_A, nr_cols_A, alpha, A,
			nr_rows_A, thrust::raw_pointer_cast(&I[0]), nr_rows_A);
	hipDeviceSynchronize();

	//get X from (L^T)X = Y
	hipblasStrsm(handle, HIPBLAS_SIDE_RIGHT, HIPBLAS_FILL_MODE_UPPER,
				HIPBLAS_OP_T,HIPBLAS_DIAG_NON_UNIT, nr_rows_A, nr_cols_A,	alpha,
				thrust::raw_pointer_cast(&L[0]), nr_rows_A, thrust::raw_pointer_cast(&I[0]), nr_rows_A);
	hipDeviceSynchronize();
	d_L.shrink_to_fit();
	L.shrink_to_fit();
	result = thrust::raw_pointer_cast(&I[0]);
}


