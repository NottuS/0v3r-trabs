#include "hip/hip_runtime.h"
#include"matrix.h"
#include <stdio.h>

//using namespace Eigen;

//Print matrix A(nr_rows_A, nr_cols_A) storage in column-major format
void print_matrix(const float *A, int nr_rows_A, int nr_cols_A) {

    for(int i = 0; i < nr_rows_A; ++i){
        for(int j = 0; j < nr_cols_A; ++j){
            std::cout << A[i * nr_rows_A + j] << " ";
        }
        std::cout << std::endl;
    }
    std::cout << std::endl;
}

__global__ void kernelCreateIdentity(float *I, int nr_rows_I){
	int id = blockIdx.x * blockDim.x + threadIdx.x;

	if (id < nr_rows_I) {
		I[id * nr_rows_I + id] = 1;
	}
}

void createIdentity(float *I, int nr_rows_I){
	dim3 dimBlock(BLOCK_START_SIZE);
	dim3 dimGrid(ceil(float(nr_rows_I)/dimBlock.x));

	kernelCreateIdentity<<<dimGrid,dimBlock>>>(I, nr_rows_I);
}

void sCreateIdentity(float *I, int nr_rows_I){
	for(int i = 0; i < nr_rows_I; i++){
		I[i*nr_rows_I + i] = 1;
	}
}

//TODO allow transpose on multiplication
void sMatMul(int transp_1, int transp_2,float *C, const float *A, const float *B,
		unsigned int hA, unsigned int wA, unsigned int wB)
{
	for (unsigned int i = 0; i < hA; ++i)
		for (unsigned int j = 0; j < wB; ++j) {
			double sum = 0;
			for (unsigned int k = 0; k < wA; ++k) {
				double a;
				double b;
				if(! transp_1)
					a = A[i * wA + k];
				else
					a = A[k * wA + i];
				if(! transp_2)
					b = B[k * wB + j];
				else
					b = B[j * wB + k];
				sum += a * b;
			}
			C[i * wB + j] = (float)sum;
		}
}

__global__ void kernelMatMul(float *C, const float *A,
		const float *B, unsigned int nr_rows_A, unsigned int nr_cols_A, unsigned int nr_cols_B){
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	__shared__ float tempA[BLOCK_SIZE][BLOCK_SIZE];
	__shared__ float tempB[BLOCK_SIZE][BLOCK_SIZE];

	float Cvalue = 0;

	if (row < nr_rows_A && col < nr_cols_B) {
		//TODO Check if all their work is done, and the indexs
		for (int i = 0; i < ceil(float(nr_rows_A)/(BLOCK_SIZE)); ++i)
		{
			/*tempA[threadIdx.y*BLOCK_SIZE + threadIdx.x] = A[row * nr_col_A + i * BLOCK_START_SIZE + threadIdx.x];
			tempB[threadIdx.y*BLOCK_SIZE + threadIdx.x] = B[(i * BLOCK_START_SIZE + threadIdx.x)* nr_cols_B *  + row];*/
			tempA[threadIdx.y][threadIdx.x] = A[row * nr_cols_A + BLOCK_SIZE * i + threadIdx.x];
			tempB[threadIdx.y][threadIdx.x] = B[(BLOCK_SIZE * i + threadIdx.y) * nr_cols_B + col];
			__syncthreads();

			#pragma unroll
			for (int j = 0; j < BLOCK_SIZE; ++j)
				Cvalue += tempA[threadIdx.y][j] * tempB[j][threadIdx.x];

			__syncthreads();
		}
		C[row * nr_cols_A + col] = Cvalue;
		
	}
}

__global__ void transp1kernelMatMul(float *C, const float *A,
		const float *B, unsigned int nr_rows_A, unsigned int nr_cols_A, unsigned int nr_cols_B){
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	__shared__ float tempA[BLOCK_SIZE][BLOCK_SIZE];
	__shared__ float tempB[BLOCK_SIZE][BLOCK_SIZE];

	float Cvalue = 0;

	if (row < nr_rows_A && col < nr_cols_B) {
		//TODO Check if all their work is done, and the indexs
		for (int i = 0; i < ceil(float(nr_rows_A)/(BLOCK_SIZE)); ++i)
		{
			/*tempA[threadIdx.y*BLOCK_SIZE + threadIdx.x] = A[row * nr_col_A + i * BLOCK_START_SIZE + threadIdx.x];
			tempB[threadIdx.y*BLOCK_SIZE + threadIdx.x] = B[(i * BLOCK_START_SIZE + threadIdx.x)* nr_cols_B *  + row];*/
			tempA[threadIdx.y][threadIdx.x] = A[(BLOCK_SIZE * i + threadIdx.y) * nr_cols_A + col];
			tempB[threadIdx.y][threadIdx.x] = B[(BLOCK_SIZE * i + threadIdx.y) * nr_cols_B + col];
			__syncthreads();

			#pragma unroll
			for (int j = 0; j < BLOCK_SIZE; ++j)
				Cvalue += tempA[j][threadIdx.y] * tempB[j][threadIdx.x];

			__syncthreads();
		}
		C[row * nr_cols_A + col] = Cvalue;

	}
}

__global__ void transp2kernelMatMul(float *C, const float *A,
		const float *B, unsigned int nr_rows_A, unsigned int nr_cols_A, unsigned int nr_cols_B){
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	__shared__ float tempA[BLOCK_SIZE][BLOCK_SIZE];
	__shared__ float tempB[BLOCK_SIZE][BLOCK_SIZE];

	float Cvalue = 0;

	if (row < nr_rows_A && col < nr_cols_B) {
		//TODO Check if all their work is done, and the indexs
		for (int i = 0; i < ceil(float(nr_rows_A)/(BLOCK_SIZE)); ++i)
		{
			/*tempA[threadIdx.y*BLOCK_SIZE + threadIdx.x] = A[row * nr_col_A + i * BLOCK_START_SIZE + threadIdx.x];
			tempB[threadIdx.y*BLOCK_SIZE + threadIdx.x] = B[(i * BLOCK_START_SIZE + threadIdx.x)* nr_cols_B *  + row];*/
			tempA[threadIdx.y][threadIdx.x] = A[row * nr_cols_A + BLOCK_SIZE * i + threadIdx.x];
			tempB[threadIdx.y][threadIdx.x] = B[row * nr_cols_B + BLOCK_SIZE * i + threadIdx.x];
			__syncthreads();

			#pragma unroll
			for (int j = 0; j < BLOCK_SIZE; ++j)
				Cvalue += tempA[threadIdx.y][j] * tempB[threadIdx.x][j] ;

			__syncthreads();
		}
		C[row * nr_cols_A + col] = Cvalue;

	}
}

//TODO allow transpose on multiplication
void pMatMul(int transp_1, int transp_2, float *C, const float *A, const float *B,
		unsigned int nr_rows_A, unsigned int nr_cols_A, unsigned int nr_cols_B){
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid(ceil(float(nr_rows_A) / dimBlock.x), ceil(float(nr_cols_B) / dimBlock.y));

	//hipSetDeviceFlags(hipDeviceLmemResizeToMax);
	if(transp_1)
		transp1kernelMatMul<<<dimGrid, dimBlock>>>(C, A, B, nr_rows_A, nr_cols_A, nr_cols_B);
	else if(transp_2)
		transp2kernelMatMul<<<dimGrid, dimBlock>>>(C, A, B, nr_rows_A, nr_cols_A, nr_cols_B);
	else
		kernelMatMul<<<dimGrid, dimBlock>>>(C, A, B, nr_rows_A, nr_cols_A, nr_cols_B);
	hipError_t e = hipGetLastError();
	if (e != hipSuccess){
		printf("%s \n", hipGetErrorString(e));
	}
}

void cublasMatMul(hipblasHandle_t &handle, int transp_1, int transp_2, float *C,
		const float *A, const float *B, unsigned int m, unsigned int k, unsigned int n){
	const float alf = 1;
	const float bet = 0;
	const float *alpha = &alf;
	const float *beta = &bet;

	// Do the actual multiplication
	// matrix - matrix multiplication : C = alf*A*B + bet*C
	// A -mxk matrix , B -kxn matrix , C -mxn matrix ;
	CUBLAS_CHECK_RETURN(hipblasSgemm(handle, (hipblasOperation_t)transp_1, (hipblasOperation_t)transp_2,
			m, n, k, alpha, A, m, B, k, beta, C, m));
}

void sMatSum(float *C, const float *A, const float *B, int nr_rows_A, int nr_cols_A){
	for(int i = 0; i < nr_rows_A * nr_cols_A; i++){
		C[i] = A[i] + B[i];
	}

}

__global__ void kernelMatSum(float *C, const float *A, const float *B, int nr_rows_A, int nr_cols_A){
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	if (row < nr_rows_A && col < nr_cols_A) {
		C[row*nr_cols_A + col] = A[row*nr_cols_A + col] + B[row*nr_cols_A + col];
	}
}

void pMatSum(float *C, const float *A, const float *B, int nr_rows_A, int nr_cols_A){
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid(ceil(float(nr_rows_A) / dimBlock.x), ceil(float(nr_cols_A) / dimBlock.y));

	//hipSetDeviceFlags(hipDeviceLmemResizeToMax);
	kernelMatSum<<<dimGrid, dimBlock>>>(C,A,B,nr_rows_A, nr_cols_A);
	//hipDeviceReset();
}

void cublasMatSum(hipblasHandle_t &handle, float *C, const float *A, const float *B, int nr_rows_A, int nr_cols_A){
	const float alf = 1;
	const float bet = 1;
	const float *alpha = &alf;
	const float *beta = &bet;

	CUBLAS_CHECK_RETURN(hipblasSgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, nr_rows_A, nr_cols_A, alpha,
			A, nr_rows_A, beta, B, nr_rows_A, C, nr_rows_A));
}

void sMatSub(float *C, const float *A, const float *B, int nr_rows_A, int nr_cols_A){
	for(int i = 0; i < nr_rows_A * nr_cols_A; i++){
		C[i] = A[i] - B[i];
	}
}

__global__ void kernelMatSub(float *C, const float *A, const float *B, int nr_rows_A, int nr_cols_A){
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	if (row < nr_rows_A && col < nr_cols_A) {
		C[row*nr_cols_A + col] = A[row*nr_cols_A + col] - B[row*nr_cols_A + col];
	}
}

void pMatSub(float *C, const float *A, const float *B, int nr_rows_A, int nr_cols_A){
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid(ceil(float(nr_rows_A) / dimBlock.x), ceil(float(nr_cols_A) / dimBlock.y));


	//hipSetDeviceFlags(hipDeviceLmemResizeToMax);
	kernelMatSub<<<dimGrid, dimBlock>>>(C,A,B,nr_rows_A, nr_cols_A);
	//hipDeviceReset();
}

void cublasMatSub(hipblasHandle_t &handle, float *C, const float *A, const float *B, int nr_rows_A, int nr_cols_A){
	const float alf = 1;
	const float bet = -1;
	const float *alpha = &alf;
	const float *beta = &bet;

	CUBLAS_CHECK_RETURN(hipblasSgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, nr_rows_A, nr_cols_A, alpha,
			A, nr_rows_A, beta, B, nr_rows_A, C, nr_rows_A));
}

void sMatTranspose(float *C, const float *A, int nr_rows_A, int nr_cols_A){
	for (int i = 0; i < nr_cols_A; ++i) {
		for (int j = 0; j < nr_rows_A; ++j) {
			C[i * nr_cols_A + j] = A[j * nr_rows_A + i];
		}
	}
}

__global__ void kernelTranspose(float *C, const float *A, int nr_rows_A, int nr_cols_A){
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	__shared__ float temp[BLOCK_SIZE][BLOCK_SIZE+1];

	if (row < nr_rows_A && col < nr_cols_A) {
		int id_in = row * nr_cols_A + col;
		temp[threadIdx.y][threadIdx.x] = A[id_in];
	}

	__syncthreads();

	col = blockIdx.y * blockDim.y + threadIdx.y;
	row = blockIdx.x * blockDim.x + threadIdx.x;

	if (row < nr_cols_A && col < nr_rows_A) {
		int id_out = col * nr_rows_A + row;
		C[id_out] = temp[threadIdx.x][threadIdx.y];
	}
}

void pMatTranspose(float *C, const float *A, int nr_rows_A, int nr_cols_A){
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid(ceil(float(nr_rows_A) / dimBlock.x), ceil(float(nr_cols_A) / dimBlock.y));

	kernelTranspose<<<dimGrid, dimBlock>>>(C, A, nr_rows_A, nr_cols_A);
}

void cublasMatTranspose(hipblasHandle_t &handle, float *C, const float *A, const float *B, int nr_rows_A, int nr_cols_A){
	const float alf = 1;
	const float bet = 0;
	const float *alpha = &alf;
	const float *beta = &bet;

	CUBLAS_CHECK_RETURN(hipblasSgeam(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, nr_rows_A, nr_cols_A, alpha,
			A, nr_rows_A, beta, B, nr_rows_A, C, nr_rows_A));
}



void sMatInverse(float *A, int nr_rows_A, int nr_cols_A, float *resultado){
	/*thrust::device_vector<float> I(nr_rows_A * nr_cols_A);
	thrust::host_vector<float> L(nr_rows_A * nr_cols_A);
	thrust::fill(I.begin(), I.end(), 0);
	hipDeviceSynchronize();
	createIdentity(thrust::raw_pointer_cast(&I[0]), nr_rows_A);
	hipDeviceSynchronize();*/

	//get L from A = LL^T
	/*choleskyDecomp(A, thrust::raw_pointer_cast(&L[0]), nr_rows_A, nr_cols_A);
*/
	gsl_matrix_float_view gA = gsl_matrix_float_view_array(A, nr_rows_A, nr_cols_A);
	//gsl_matrix_float *gA = gsl_matrix_float_alloc (nr_rows_A, nr_cols_A);
	//gsl_linalg_float_cholesky_decomp(&gA.matrix);
	//gsl_linalg_float_cholesky_invert(&gA.matrix);
}

void choleskyDecomp(const float *A, float *L, int nr_rows_A, int nr_cols_A){
	int i,j,k;
	float sum;
	for(i = 0; i < nr_rows_A; i++) {
		for (j = 0; j <= i; ++j) {
			sum = A[i*nr_cols_A + j];
			for(k = 0; k < j; k++) {
				sum -= L[i*nr_cols_A +k] * L[j*nr_cols_A + k];
			}
			if(i == j) {
				L[i*nr_cols_A + j] = sqrt(sum);
			} else {
				L[i*nr_cols_A + j] = sum / L[j*nr_cols_A + j];
			}
		}
	}
}

__global__ void choleskyDecompKernel2(int ind, const float *A, float *L, int nr_rows_A, int nr_cols_A){
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	__shared__ float tempA[BLOCK_SIZE][BLOCK_SIZE];
	__shared__ float tempB[BLOCK_SIZE][BLOCK_SIZE];

	if (row < nr_rows_A && col < ind) {
		//TODO Check if all their work is done, and the indexs
		float sum = A[row * nr_cols_A + ind];

		for (int i = 0; i < ceil(float(ind)/(BLOCK_SIZE)); ++i) {

			/*tempA[threadIdx.y*BLOCK_SIZE + threadIdx.x] = A[row * nr_col_A + i * BLOCK_START_SIZE + threadIdx.x];
			tempB[threadIdx.y*BLOCK_SIZE + threadIdx.x] = B[(i * BLOCK_START_SIZE + threadIdx.x)* nr_cols_B *  + row];*/
			tempA[threadIdx.y][threadIdx.x] = A[ind * nr_cols_A + BLOCK_SIZE * i + threadIdx.x];
			tempB[threadIdx.y][threadIdx.x] = A[row * nr_cols_A + BLOCK_SIZE * i + threadIdx.x];
			__syncthreads();
			if(ind > (i + 1) * BLOCK_SIZE){
				for (int j = 0; j < BLOCK_SIZE ; ++j)
					sum -= tempA[threadIdx.y][j] * tempB[threadIdx.y][j];
			} else {
				for (int j = 0; j < ind % BLOCK_SIZE ; ++j)
					sum -= tempA[threadIdx.y][j] * tempB[threadIdx.y][j];
			}
			__syncthreads();
		}

		if( row == ind )
			sum = sqrtf(sum);
		L[row * nr_cols_A + ind] = sum;
	}
}

__global__ void choleskyDecompKernel(int ind, const float *A, float *L, int nr_rows_A, int nr_cols_A){
	/*int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;*/
	int row = blockIdx.x * blockDim.x + threadIdx.x;
	
	__shared__ float temp1[BLOCK_START_SIZE];
	//__shared__ float temp2[BLOCK_START_SIZE];
	
	if (row < nr_rows_A) {
		int x = threadIdx.x;
		float sum = A[row * nr_cols_A + ind];
		for (int k = 0; k < ceilf((ind + 0.0)/BLOCK_START_SIZE); ++k) {
			temp1[x] = L[ind * nr_cols_A + BLOCK_START_SIZE * k + x];
			//temp2[x] = L[row * nr_cols_A + BLOCK_START_SIZE * k + 0];

			__syncthreads( );
			if((k+1)*BLOCK_START_SIZE <= ind) {
				#pragma unroll
				for(int i = 0; i < BLOCK_START_SIZE/8; i += 8){
					float a,b,c,d, a1, b1, c1, d1;
					a = L[row * nr_cols_A + k * BLOCK_START_SIZE + i];
					b = L[row * nr_cols_A + k * BLOCK_START_SIZE + i + 1];
					c = L[row * nr_cols_A + k * BLOCK_START_SIZE + i + 2];
					d = L[row * nr_cols_A + k * BLOCK_START_SIZE + i + 3];
					a1 = L[row * nr_cols_A + k * BLOCK_START_SIZE + i + 4];
					b1 = L[row * nr_cols_A + k * BLOCK_START_SIZE + i + 5];
					c1 = L[row * nr_cols_A + k * BLOCK_START_SIZE + i + 6];
					d1 = L[row * nr_cols_A + k * BLOCK_START_SIZE + i + 7];
					//sum -= L[row * nr_cols_A + k * BLOCK_START_SIZE + i] * temp1[i];
					//sum -= temp2[i] * temp1[i];
					sum -= a * temp1[i] + b * temp1[i+1] + c * temp1[i+2] + d * temp1[i+3] +
							a1 * temp1[i+4] + b1 * temp1[i+5] + c1 * temp1[i+6] + d1 * temp1[i+7];
				}
			} else {
				for(int i = 0; i < ind % BLOCK_START_SIZE/8; i += 8){
					float a,b,c,d, a1, b1, c1, d1;
					a = L[row * nr_cols_A + k * BLOCK_START_SIZE + i];
					b = L[row * nr_cols_A + k * BLOCK_START_SIZE + i + 1];
					c = L[row * nr_cols_A + k * BLOCK_START_SIZE + i + 2];
					d = L[row * nr_cols_A + k * BLOCK_START_SIZE + i + 3];
					a1 = L[row * nr_cols_A + k * BLOCK_START_SIZE + i + 4];
					b1 = L[row * nr_cols_A + k * BLOCK_START_SIZE + i + 5];
					c1 = L[row * nr_cols_A + k * BLOCK_START_SIZE + i + 6];
					d1 = L[row * nr_cols_A + k * BLOCK_START_SIZE + i + 7];
					//sum -= L[row * nr_cols_A + k * BLOCK_START_SIZE + i] * temp1[i];
					//sum -= temp2[i] * temp1[i];
					sum -= a * temp1[i] + b * temp1[i+1] + c * temp1[i+2] + d * temp1[i+3] +
							a1 * temp1[i+4] + b1 * temp1[i+5] + c1 * temp1[i+6] + d1 * temp1[i+7];
				}
			}
			__syncthreads();
		}

		if( row == ind )
			sum = sqrtf(sum);
		L[row * nr_cols_A + ind] = sum;
	}
}

__global__ void updateCholesky(int ind, float *L, int nr_rows_A, int nr_cols_A){
	int row = blockIdx.x * blockDim.x + threadIdx.x;
	
	if (row < nr_rows_A && row != ind ) {
		if( row < ind )
			L[row * nr_cols_A + ind] = 0;
		else
			L[row * nr_cols_A + ind] /=  L[ind * nr_cols_A + ind];
	}
}

//TODO
void pMatInverse(const float *A, float *L, int nr_rows_A, int nr_cols_A){
	dim3 dimBlock(BLOCK_START_SIZE);
	dim3 dimBlock2(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid(ceil(float(nr_rows_A) / dimBlock.x));
	dim3 dimGrid2(ceil(float(nr_rows_A) / dimBlock.x), ceil(float(nr_cols_A) / dimBlock.y));

	for(int i = 0; i < nr_rows_A; i++){
		choleskyDecompKernel<<<dimGrid, dimBlock>>>(i, A, L, nr_rows_A, nr_cols_A);
		CUDA_CHECK_RETURN(hipDeviceSynchronize());
		updateCholesky<<<dimGrid, dimBlock>>>(i, L, nr_rows_A, nr_cols_A);
		CUDA_CHECK_RETURN(hipDeviceSynchronize());
	}
}

//TODO correct this one
void cublasMatInverse(hipblasHandle_t &handle, const float *A, int nr_rows_A, int nr_cols_A, float *result){

	const float alf = 1;
	const float *alpha = &alf;

	//create the identity matrix
	thrust::device_vector<float> I(nr_rows_A * nr_cols_A);
	thrust::host_vector<float> L(nr_rows_A * nr_cols_A);
	thrust::fill(I.begin(), I.end(), 0);
	hipDeviceSynchronize();
	createIdentity(thrust::raw_pointer_cast(&I[0]), nr_rows_A);
	hipDeviceSynchronize();

	//get L from A = LL^T
	choleskyDecomp(A, thrust::raw_pointer_cast(&L[0]), nr_rows_A, nr_cols_A);
	thrust::device_vector<float> d_L(L.begin(), L.begin() + L.size());

	//get Y from LY = I
	hipblasStrsm(handle,HIPBLAS_SIDE_RIGHT, HIPBLAS_FILL_MODE_LOWER,
			HIPBLAS_OP_N,HIPBLAS_DIAG_NON_UNIT, nr_rows_A, nr_cols_A, alpha, A,
			nr_rows_A, thrust::raw_pointer_cast(&I[0]), nr_rows_A);
	hipDeviceSynchronize();

	//get X from (L^T)X = Y
	hipblasStrsm(handle, HIPBLAS_SIDE_RIGHT, HIPBLAS_FILL_MODE_UPPER,
				HIPBLAS_OP_T,HIPBLAS_DIAG_NON_UNIT, nr_rows_A, nr_cols_A,	alpha,
				thrust::raw_pointer_cast(&L[0]), nr_rows_A, thrust::raw_pointer_cast(&I[0]), nr_rows_A);
	hipDeviceSynchronize();
	d_L.shrink_to_fit();
	L.shrink_to_fit();
	result = thrust::raw_pointer_cast(&I[0]);
}


