#include"matrix.h"
#include <stdio.h>

//Print matrix A(nr_rows_A, nr_cols_A) storage in column-major format
void print_matrix(const float *A, int nr_rows_A, int nr_cols_A) {

    for(int i = 0; i < nr_rows_A; ++i){
        for(int j = 0; j < nr_cols_A; ++j){
            std::cout << A[j * nr_rows_A + i] << " ";
        }
        std::cout << std::endl;
    }
    std::cout << std::endl;
}

void sMatMul(float *C, const float *A, const float *B,
		unsigned int hA, unsigned int wA, unsigned int wB)
{
	for (unsigned int i = 0; i < hA; ++i)
		for (unsigned int j = 0; j < wB; ++j) {
			double sum = 0;
			for (unsigned int k = 0; k < wA; ++k) {
				double a = A[i * wA + k];
				double b = B[k * wB + j];
				sum += a * b;
			}
			C[i * wB + j] = (float)sum;
		}
}

void pMatMul(float *C, const float *A, const float *B, unsigned int hA, unsigned int wA, unsigned int wB){

}

void cublasMatMul(hipblasHandle_t &handle, float *C,
		const float *A, const float *B, unsigned int m, unsigned int k, unsigned int n){
	const float alf = 1;
	const float bet = 0;
	const float *alpha = &alf;
	const float *beta = &bet;

	// Do the actual multiplication
	// matrix - matrix multiplication : C = alf*A*B + bet*C
	// A -mxk matrix , B -kxn matrix , C -mxn matrix ;
	CUBLAS_CHECK_RETURN(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
			m, n, k, alpha, A, m, B, k, beta, C, m));
}

void pMatSum(const float *C, const float *A, const float *B, int nr_rows_A, int nr_cols_A){

}

void cublasMatSum(hipblasHandle_t &handle, const float *C, const float *A, const float *B, int nr_rows_A, int nr_cols_A){
	const float alf = 1;
	const float bet = 1;
	const float *alpha = &alf;
	const float *beta = &bet;

	CUBLAS_CHECK_RETURN(hipblasSgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, nr_rows_A, nr_cols_A, alpha,
			A, nr_rows_A, B, nr_rows_A, beta, C, nr_rows_A));
}

void pMatSub(const float *C, const float *A, const float *B, int nr_rows_A, int nr_cols_A){

}

void cublasMatSub(hipblasHandle_t &handle, const float *C, const float *A, const float *B, int nr_rows_A, int nr_cols_A){
	const float alf = 1;
	const float bet = -1;
	const float *alpha = &alf;
	const float *beta = &bet;

	CUBLAS_CHECK_RETURN(hipblasSgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, nr_rows_A, nr_cols_A, alpha,
			A, nr_rows_A, B, nr_rows_A, beta, C, nr_rows_A));
}

void pMatTranspose(const float *A, int nr_rows_A, int nr_cols_A){

}

void cublasMatTranspose(hipblasHandle_t &handle, const float *C, const float *A, const float *B, int nr_rows_A, int nr_cols_A){
	const float alf = 1;
	const float bet = 0;
	const float *alpha = &alf;
	const float *beta = &bet;

	CUBLAS_CHECK_RETURN(hipblasSgeam(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, nr_rows_A, nr_cols_A, alpha,
			A, nr_rows_A, B, nr_rows_A, beta, C, nr_rows_A));
}

void sInvMat(const float *A, int nr_rows_A, int nr_cols_A){

}
void pInvMat(const float *A, int nr_rows_A, int nr_cols_A){

}
void cublasInvMat(hipblasHandle_t &handle, const float *A, int nr_rows_A, int nr_cols_A){

}

